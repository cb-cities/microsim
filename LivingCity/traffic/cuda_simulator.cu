#include "hip/hip_runtime.h"
// CUDA CODE
#include "assert.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>

#include "cuda_simulator.h"

#include <iostream>
#include <random>

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

///////////////////////////////
// CONSTANTS

__constant__ float intersectionClearance = 7.8f;

using namespace LC;
////////////////////////////////
// VARIABLES
LC::Agent *trafficPersonVec_d;
uint *indexPathVec_d;
LC::EdgeData *edgesData_d;
LC::IntersectionData *intersections_d;
uchar *laneMap_d;

__managed__ bool readFirstMapC = true;
__managed__ uint mapToReadShift;
__managed__ uint mapToWriteShift;
__managed__ int mutex = 0;
__managed__ uint halfLaneMap;

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n",
           hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double)free_byte;
  double total_db = (double)total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n",
         used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0,
         total_db / 1024.0 / 1024.0);
}

//! Allocate appropirate amount of memory on the cuda device
void init_cuda(bool fistInitialization, // create buffers
               std::vector<LC::Agent> &agents,
               std::vector<LC::EdgeData> &edgesData,
               std::vector<uchar> &laneMap,
               std::vector<LC::IntersectionData> &intersections) {

  { // agents
    size_t size = agents.size() * sizeof(LC::Agent);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&trafficPersonVec_d,
                           size)); // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, agents.data(), size,
                         hipMemcpyHostToDevice));
  }

  { // edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::EdgeData);
    if (fistInitialization)
      gpuErrchk(
          hipMalloc((void **)&edgesData_d, sizeD)); // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD,
                         hipMemcpyHostToDevice));
  }
  { // laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization)
      gpuErrchk(
          hipMalloc((void **)&laneMap_d, sizeL)); // Allocate array on device
    gpuErrchk(
        hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  { // intersections
    size_t sizeI = intersections.size() * sizeof(LC::IntersectionData);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&intersections_d,
                           sizeI)); // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, intersections.data(), sizeI,
                         hipMemcpyHostToDevice));
  }
  printMemoryUsage();
} //

//! free gpu memories
void finish_cuda(void) {
  //////////////////////////////
  // FINISH
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
} //

void cuda_get_data(std::vector<LC::Agent> &trafficPersonVec,
                   std::vector<LC::EdgeData> &edgesData,
                   std::vector<LC::IntersectionData> &intersections) {
  // copy back people
  size_t size = trafficPersonVec.size() * sizeof(LC::Agent);
  size_t size_edges = edgesData.size() * sizeof(LC::EdgeData);
  size_t size_intersections =
      intersections.size() * sizeof(LC::IntersectionData);

  hipMemcpy(trafficPersonVec.data(), trafficPersonVec_d, size,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
  hipMemcpy(edgesData.data(), edgesData_d, size_edges,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
  hipMemcpy(intersections.data(), intersections_d, size_intersections,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
}

__device__ uint lanemap_pos(const uint currentEdge, const uint edge_length,
                            const uint laneNum, const uint pos_in_lane) {
  uint kMaxMapWidthM = 1024;
  uint num_cell = pos_in_lane / kMaxMapWidthM;
  int tot_num_cell = edge_length / kMaxMapWidthM;
  if (edge_length % kMaxMapWidthM) {
    tot_num_cell += 1;
  }
  return kMaxMapWidthM * currentEdge + kMaxMapWidthM * laneNum * tot_num_cell +
         kMaxMapWidthM * num_cell + pos_in_lane % kMaxMapWidthM;
}

__device__ void calculateGaps(uchar *laneMap, LC::Agent &agent,
                              uint laneToCheck, float &gap_a, float &gap_b,
                              uchar &v_a, uchar &v_b) {

  // CHECK FORWARD
  for (ushort b = agent.posInLaneM - 1; b < agent.edge_length;
       b++) { // NOTE -1 to make sure there is none in at the same level
    auto posToSample =
        lanemap_pos(agent.edge_mid, agent.edge_length, laneToCheck, b);
    if (laneMap[mapToReadShift + posToSample] != 0xFF) {
      gap_a = b - agent.posInLaneM; // m
      v_a = laneMap[mapToReadShift + posToSample] / 3;
      break;
    }
  }
  // CHECK BACKWARD
  for (ushort b = agent.posInLaneM + 1; b > 0;
       b--) { // NOTE -1 to make sure there is none in at the same level
    auto posToSample =
        lanemap_pos(agent.edge_mid, agent.edge_length, laneToCheck, b);
    if (laneMap[mapToReadShift + posToSample] != 0xFF) {
      gap_b = agent.posInLaneM - b; // m
      v_b = laneMap[mapToReadShift + posToSample] / 3;
      break;
    }
  }
}

// TODO : CHECK MULTIPLE LANES
__device__ bool check_space(int space, int eid, int edge_length, uchar *laneMap,
                            uint mapToReadShift) {
  for (auto b = 0; b < space; b++) {
    // just right LANE !!!!!!!
    auto pos = lanemap_pos(eid, edge_length, 0, b);
    auto laneChar =
        laneMap[mapToReadShift + pos]; // get byte of edge (proper line)
    if (laneChar != 0xFF) {
      return false;
    }
  }
  return true;
}

__device__ int deque(int *queue, unsigned &rear) {
  int aid = queue[0];
  for (int i = 0; i < rear - 1; i++) {
    queue[i] = queue[i + 1];
  }
  rear--; // decrement rear
  return aid;
}

__device__ void initialize_agent(int agent_id, LC::Agent &agent,
                                 LC::EdgeData *edgesData, uchar *laneMap,
                                 LC::IntersectionData *intersections) {

  // 1.1  edge case: no available route
  if (agent.route_size == 0) {
    agent.active = 2;
    return;
  }
  // add to corresponding queue
  auto &intersection = intersections[agent.init_intersection];
  intersection.init_queue[intersection.init_queue_rear] = agent_id;
  atomicAdd(&(intersection.init_queue_rear), 1);

  // initialize agent
  agent.active = 1;

  //        bool isSet = false;
  //        do {
  //          if (isSet = atomicCAS(&mutex0, 0, 1) == 0) {
  //            intersection.init_queue[intersection.init_queue_rear] =
  //            agent_id; atomicAdd(&(intersection.init_queue_rear), 1);
  //          }
  //          if (isSet) {
  //            atomicExch(&mutex0, 0);
  //            __syncthreads();
  //          }
  //        } while (!isSet);
}

// TODO : CHECK NEXT EDGE?
__device__ void check_front_car(LC::Agent &agent, uchar *laneMap,
                                float deltaTime) {

  int numCellsCheck = fmax(15.0f, agent.v * deltaTime); // 15 or speed*time
  ushort byteInLine = (ushort)floor(agent.posInLaneM);

  // a) SAME LINE (BEFORE SIGNALING)
  float s = 20;
  float delta_v = agent.v - agent.max_speed;
  for (ushort b = byteInLine + 1;
       (b < agent.edge_length) && (numCellsCheck > 0); b++, numCellsCheck--) {
    uint posToSample =
        lanemap_pos(agent.edge_mid, agent.edge_length, agent.lane, b);
    auto laneChar = laneMap[mapToReadShift + posToSample];
    if (laneChar != 0xFF) {
      s = ((float)(b - byteInLine)); // m
      delta_v =
          agent.v -
          (laneChar / 3.0f); // laneChar is in 3*ms (to save space in array)
      break;
    }
  }
  agent.s = s;
  agent.delta_v = delta_v;

  // NEXT LINE
  // e) MOVING ALONG IN THE NEXT EDGE
  //  if (!found && numCellsCheck > 0) { // check if in next line
  //    if ((agent.nextEdge != -1)) {    // we haven't arrived to
  //      // destination next line)
  //      ushort nextEdgeLaneToBe = agent.lane; // same lane
  //
  //      // printf("trafficPersonVec[p].lane
  //      // %u\n",trafficPersonVec[p].lane);
  //      if (nextEdgeLaneToBe >= agent.nextEdgeNumLanes) {
  //        nextEdgeLaneToBe =
  //            agent.nextEdgeNumLanes - 1; // change line if there are less
  //            roads
  //      }
  //
  //      ushort numOfCells = ceil(agent.nextEdgeLength);
  //
  //      for (ushort b = 0; (b < numOfCells) && (numCellsCheck > 0);
  //           b++, numCellsCheck--) {
  //        // laneChar = laneMap[mapToReadShift + maxWidth * (nextEdge +
  //        // nextEdgeLaneToBe) + b];
  //        uint posToSample = lanemap_pos(agent.nextEdge, nextEdgeLaneToBe, b);
  //        auto laneChar = laneMap[mapToReadShift + posToSample];
  //
  //        if (laneChar != 0xFF) {
  //          s = ((float)(b)); // m
  //          delta_v = agent.v -
  //                    (laneChar / 3.0f); // laneChar is in 3*ms (to save space
  //                    in
  //          break;
  //        }
  //      }
  //    }
  //  }
}

__device__ void update_agent_info(LC::Agent &agent, float deltaTime) {

  // update speed
  float thirdTerm = 0;
  if (agent.delta_v > -0.01) { // car in front and slower than us
    // 2.1.2 calculate dv_dt
    float s_star =
        agent.s_0 +
        fmax(0.0f, (agent.v * agent.T + (agent.v * agent.delta_v) /
                                            (2 * sqrtf(agent.a * agent.b))));

    thirdTerm = powf(((s_star) / (agent.s)), 2);
    agent.slow_down_steps++;
  }
  float dv_dt =
      agent.a * (1.0f - std::pow((agent.v / agent.max_speed), 4) - thirdTerm);
  // 2.1.3 update values
  float numMToMove =
      fmax(0.0f, agent.v * deltaTime + 0.5f * (dv_dt)*deltaTime * deltaTime);
  agent.v += dv_dt * deltaTime;
  agent.dv_dt = dv_dt;
  if (agent.v < 0 or
      (agent.s - numMToMove < SOCIAL_DIST and agent.v - agent.delta_v < 0.1)) {
    agent.v = 0;
    numMToMove = 0;
  }
  agent.cum_length += numMToMove;
  agent.cum_v += agent.v;
  agent.posInLaneM += numMToMove;
}

__device__ void change_lane(LC::Agent &agent, LC::EdgeData *edgesData,
                            uchar *laneMap) {

  auto &current_edge = edgesData[agent.edge_mid];
  if (agent.posInLaneM > current_edge.length) { // skip if will go to next edge
    return;
  }
  if (current_edge.num_lanes < 2 || agent.v > 0.9 * agent.max_speed) {
    return; // skip if reach the destination/have no lane to change/cruising
            // (avoid periodic lane changing)
  }

  if (agent.v > 3.0f &&           // at least 10km/h to try to change lane
      agent.delta_v > -0.1 &&     // decelerating or stuck
      agent.num_steps % 5 == 0) { // just check every (5 steps) 5 seconds

    bool leftLane = agent.lane > 0; // at least one lane on the left
    bool rightLane =
        agent.lane < current_edge.num_lanes - 1; // at least one lane

    if (leftLane && rightLane) {
      if (int(agent.v) % 2 == 0) { // pseudo random for change lane
        rightLane = false;
      }
    }

    ushort laneToCheck = agent.lane - 1;
    if (rightLane) {
      laneToCheck = agent.lane + 1;
    }

    uchar v_a, v_b;
    float gap_a = 1000.0f, gap_b = 1000.0f;
    calculateGaps(laneMap, agent, laneToCheck, gap_a, gap_b, v_a, v_b);

    // Safe distance calculation
    float b1A = 0.05, b2A = 0.15;
    float b1B = 0.15, b2B = 0.40;
    // simParameters.s_0-> critical lead gap
    float g_na_D =
        fmax(agent.s_0, agent.s_0 + b1A * agent.v + b2A * (agent.v - v_a));
    float g_bn_D =
        fmax(agent.s_0, agent.s_0 + b1B * v_b + b2B * (v_b - agent.v));
    if (gap_b < g_bn_D || gap_a < g_na_D) { // gap smaller than critical gap
      return;
    }

    agent.lane = laneToCheck; // CHANGE LINE
    agent.num_lane_change += 1;
  }
}

__device__ uint find_intersetcion_id(LC::Agent &agent,
                                     LC::EdgeData *edgesData) {
  // find the intersection id
  auto &current_edge = edgesData[agent.edge_mid];
  auto &next_edge = edgesData[agent.route[agent.route_ptr + 1]];
  for (unsigned i = 0; i < 2; i++) {
    auto vid = current_edge.vertex[i];
    for (unsigned j = 0; j < 2; j++) {
      if (next_edge.vertex[j] == vid) {
        return vid;
      }
    }
  }
  return 0;
}

__device__ uint find_queue_id(LC::Agent &agent,
                              LC::IntersectionData &intersection) {
  for (unsigned i = 0; i < intersection.num_queue; i++) {
    if (agent.edge_mid == intersection.start_edge[i] and
        agent.route[agent.route_ptr + 1] == intersection.end_edge[i]) {
      return i;
    }
  }
  return 0;
  //  int pos0, pos1;
  //  for (unsigned i = 0; i < intersection.num_edge; i++) {
  //    if (agent.edge_ptr == intersection.lanemap_id[i]) {
  //      pos0 = i;
  //    }
  //    if (agent.nextEdge == intersection.lanemap_id[i]) {
  //      pos1 = i;
  //    }
  //  }
  //  int base_idx = 0;
  //  if (pos0 > pos1) {
  //    base_idx += intersection.num_queue;
  //    int temp = pos1;
  //    pos1 = pos0;
  //    pos0 = temp;
  //  }
  //  int idx = (pos0 * intersection.num_edge - (pos0 * (pos0 + 1)) / 2 + pos1 -
  //             pos0 - 1);
  //  return idx;
}

__device__ bool update_intersection(int agent_id, LC::Agent &agent,
                                    LC::EdgeData *edgesData,
                                    LC::IntersectionData *intersections) {
  auto &current_edge = edgesData[agent.edge_mid];
  auto extra = agent.posInLaneM - agent.edge_length;
  if (extra < 0) { // does not reach an intersection
    return false;
  }
  agent.cum_length -= extra;                     // remove the extra distance
  if (agent.route_ptr + 1 >= agent.route_size) { // reach destination
    agent.active = 2;
    atomicAdd(&(current_edge.downstream_veh_count), 1);
    int num_steps_in_edge = agent.num_steps - agent.num_steps_entering_edge;
    atomicAdd(&(current_edge.period_cum_travel_steps),
              num_steps_in_edge); // for average travel time calculation
    return false;
  }
  auto intersetcion_id = find_intersetcion_id(agent, edgesData);
  auto &intersection = intersections[intersetcion_id];
  int queue_id = find_queue_id(agent, intersection);
  auto &queue = intersection.queue[queue_id];
  auto &queue_ptr = intersection.pos[queue_id];
  agent.in_queue = true;
  agent.v = 0; // in queue vehicle is stopped.
  int num_steps_in_edge = agent.num_steps - agent.num_steps_entering_edge;
  atomicAdd(&(current_edge.period_cum_travel_steps),
            num_steps_in_edge); // for average travel time calculation

    queue[queue_ptr] = agent_id;
    atomicAdd(&(queue_ptr), 1);
    atomicAdd(&(current_edge.downstream_veh_count), 1);

  // Synchronization Control
//  bool isSet = false;
//  do {
//    if (isSet = atomicCAS(&mutex, 0, 1) == 0) {
//      queue[queue_ptr] = agent_id;
//      atomicAdd(&(queue_ptr), 1);
//      atomicAdd(&(current_edge.downstream_veh_count), 1);
//    }
//    if (isSet) {
//      atomicExch(&mutex, 0);
//      __syncthreads();
//    }
//  } while (!isSet);
  return true;
}

__device__ void write2lane_map(LC::Agent &agent, LC::EdgeData *edgesData,
                               uchar *laneMap) {
  // write to the lanemap if still on the edge

  auto posToSample = lanemap_pos(agent.edge_mid, agent.edge_length, agent.lane,
                                 agent.posInLaneM);
  uchar vInMpS = (uchar)(agent.v * 3); // speed in m/s to fit in uchar
  laneMap[mapToWriteShift + posToSample] = vInMpS;
}

//! Simulate agents movements on network edges
__global__ void
kernel_trafficSimulation(int numPeople, float currentTime, LC::Agent *agents,
                         LC::EdgeData *edgesData, uchar *laneMap,
                         LC::IntersectionData *intersections, float deltaTime) {

  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p >= numPeople) {
    return; // CUDA check (inside margins)
  }
  if (threadIdx.x == 0) {
    mutex = 0;
  }
  __syncthreads();

  auto &agent = agents[p];
  // 1. initialization
  if (agent.active == 2) { // agent is already finished
    return;
  }
  // 1.1. check if person should still wait or should start
  if (agent.active == 0) {
    if (agent.time_departure > currentTime) { // wait
      return;
    } else { // its your turn
      initialize_agent(p, agent, edgesData, laneMap, intersections);
      return;
    }
  }

  // 2. Moving
  agent.num_steps++;
  if (agent.in_queue) {
    agent.num_steps_in_queue += 1;
    return;
  }

  // 2.1.1 Find front car
  check_front_car(agent, laneMap, deltaTime);
  // 2.1.2 Update agent information using the front car info
  update_agent_info(agent, deltaTime);
  //  2.1.3 Perform lane changing if necessary
  change_lane(agent, edgesData, laneMap);
  // 2.1.4 check intersection
  bool added2queue = update_intersection(p, agent, edgesData, intersections);
  // 2.1.5 write the updated agent info to lanemap
  if (not added2queue) {
    write2lane_map(agent, edgesData, laneMap);
  }

} //

// TODO : PLACE ON MULTIPLE LANES
__device__ void move2nextEdge(LC::Agent &agent, int numMToMove,
                              LC::EdgeData *edgesData, uchar *laneMap) {

  agent.in_queue = false;
  agent.route_ptr++;
  agent.edge_mid = agent.route[agent.route_ptr];
  agent.posInLaneM = numMToMove;
  agent.lane = 0;
  agent.v = INIT_SPEED; // double initial speed to avoid unnecessary queueing

  auto &current_edge = edgesData[agent.edge_mid];
  agent.max_speed = current_edge.maxSpeedMperSec;
  agent.edge_length = current_edge.length;
  agent.num_steps_entering_edge = agent.num_steps;
  //
  atomicAdd(&(current_edge.upstream_veh_count), 1);

  auto posToSample = lanemap_pos(agent.edge_mid, current_edge.length,
                                 agent.lane, agent.posInLaneM);
  uchar vInMpS = (uchar)(agent.v * 3); // speed in m/s to fit in uchar
  laneMap[mapToWriteShift + posToSample] = vInMpS;
  //
  agent.cum_length += numMToMove;
  agent.num_steps += 1;
}

__device__ bool discharge_queue(LC::IntersectionData &intersection,
                                LC::Agent *trafficPersonVec,
                                LC::EdgeData *edgesData, uchar *laneMap) {
  auto &q1 = intersection.queue[intersection.queue_ptr];
  auto &n1 = intersection.pos[intersection.queue_ptr];
  unsigned eid1 = intersection.end_edge[intersection.queue_ptr];
  int edge_length = edgesData[eid1].length;
  unsigned numMToMove = SOCIAL_DIST;
  bool enough_space =
      check_space(numMToMove + SOCIAL_DIST, eid1, edge_length, laneMap,
                  mapToReadShift); // check social dist ahead
  intersection.max_queue = max(intersection.max_queue, n1);
  if (enough_space) {
    auto aid = deque(q1, n1);
    auto &agent = trafficPersonVec[aid];
    move2nextEdge(agent, numMToMove, edgesData,
                  laneMap); // move to the next edge
    return true;
  }
  return false;
}

__device__ void place_stop(LC::Agent &agent, LC::EdgeData *edgesData,
                           uchar *laneMap, uint mapToWriteShift) {
  auto &edge = edgesData[agent.edge_mid];
  for (int j = 0; j < SOCIAL_DIST; ++j) {
    auto pos = agent.edge_length - j;
    for (int i = 0; i < edge.num_lanes; ++i) {
      auto posToSample = lanemap_pos(agent.edge_mid, agent.edge_length, i, pos);
      laneMap[mapToWriteShift + posToSample] = 0;
    }
  }
}

__device__ void discharge_init_agents(unsigned intersection_id,
                                      LC::EdgeData *edgesData,
                                      LC::IntersectionData *intersections,
                                      LC::Agent *trafficPersonVec,
                                      uchar *laneMap) {
  auto &intersection = intersections[intersection_id];
  auto &init_queue = intersection.init_queue;
  auto &rear_ptr = intersection.init_queue_rear;
  if (rear_ptr > 0) {
    auto aid = init_queue[0];
    auto &agent = trafficPersonVec[aid];
    auto enough_space =
        check_space(SOCIAL_DIST+1, agent.edge_mid, agent.edge_length, laneMap,
                    mapToReadShift); // check social dist ahead
    if (enough_space) {
      aid = deque(init_queue, rear_ptr);
      agent.posInLaneM = SOCIAL_DIST; // start from social dist
      agent.v = INIT_SPEED;
      agent.edge_mid = agent.route[agent.route_ptr];
      auto &current_edge = edgesData[agent.edge_mid];
      agent.edge_length = current_edge.length;
      agent.max_speed = current_edge.maxSpeedMperSec;

      uchar vInMpS =
          (uchar)(agent.v * 3); // speed in m/s *3 (to keep more precisio
      auto pos = lanemap_pos(agent.edge_mid, agent.edge_length, agent.lane,
                             agent.posInLaneM);
      laneMap[mapToWriteShift + pos] = vInMpS;
      agent.cum_length += agent.posInLaneM;
      agent.num_steps += 1;

      atomicAdd(&(current_edge.upstream_veh_count), 1);
    }
    // update waiting steps for all other agents
    for (int i = 0; i < rear_ptr; ++i) {
      auto aid = init_queue[i];
      auto &agent = trafficPersonVec[aid];
      agent.initial_waited_steps += 1;
    }
  }
}

__device__ void check_queues(unsigned intersection_id, LC::EdgeData *edgesData,
                             LC::IntersectionData *intersections,
                             LC::Agent *trafficPersonVec, uchar *laneMap) {
  auto &intersection = intersections[intersection_id];
  auto &queue_counter = intersection.pos;
  unsigned start_ptr = intersection.queue_ptr;

  while (intersection.queue_ptr + 1 != start_ptr) {
    if (queue_counter[intersection.queue_ptr] > 0) {
      bool discharged =
          discharge_queue(intersection, trafficPersonVec, edgesData, laneMap);

      if (discharged) { // if queue cleared, go to the next queue
        if (intersection.queue_ptr + 1 < intersection.num_queue) {
          intersection.queue_ptr += 1;
        } else {
          intersection.queue_ptr = 0;
          if (start_ptr == 0)
            break;
        }
      }
      break;
    } else { // no car in the queue, go to the next state
      if (intersection.queue_ptr + 1 < intersection.num_queue) {
        intersection.queue_ptr += 1;
      } else {
        intersection.queue_ptr = 0;
        discharge_init_agents(intersection_id, edgesData, intersections,
                              trafficPersonVec, laneMap); // roundrobin
        if (start_ptr == 0)
          break;
      }
    }
  }
}

//! Simulate agents movements on intersections
__global__ void
kernel_intersectionOneSimulation(uint numIntersections, LC::EdgeData *edgesData,
                                 LC::IntersectionData *intersections,
                                 LC::Agent *agents, uchar *laneMap) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numIntersections) {
    return; // CUDA check (inside margins)
  }
  check_queues(i, edgesData, intersections, agents, laneMap);

  // add a stop sign for full queues
  auto &intersection = intersections[i];
  for (unsigned j = 0; j < intersection.num_queue; j++) {
    auto num_cars = intersection.pos[j];
    if (num_cars > 0) {
      auto &q1 = intersection.queue[j];
      auto &agent = agents[q1[0]];
      place_stop(agent, edgesData, laneMap, mapToWriteShift);
    }
  }
}

void cuda_simulate(float currentTime, uint numPeople, uint numIntersections,
                   float deltaTime, int numBlocks, int threadsPerBlock) {

  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if (readFirstMapC) {
    mapToReadShift = 0;
    mapToWriteShift = halfLaneMap;
    gpuErrchk(
        hipMemset(&laneMap_d[halfLaneMap], -1,
                   halfLaneMap * sizeof(unsigned char))); // clean second half
  } else {
    mapToReadShift = halfLaneMap;
    mapToWriteShift = 0;
    gpuErrchk(
        hipMemset(&laneMap_d[0], -1,
                   halfLaneMap * sizeof(unsigned char))); // clean first half
  }
  readFirstMapC = !readFirstMapC; // next iteration invert use

  std::random_device
      rd; // Will be used to obtain a seed for the random number engine
  std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  std::uniform_int_distribution<> random_bool(0, 1);
  // random assign which to go
  intersectionBench.startMeasuring();
  kernel_intersectionOneSimulation<<<numBlocks, threadsPerBlock>>>(
      numIntersections, edgesData_d, intersections_d, trafficPersonVec_d,
      laneMap_d);
  gpuErrchk(hipPeekAtLastError());
  intersectionBench.stopMeasuring();

  peopleBench.startMeasuring();
  // Simulate people.
  kernel_trafficSimulation<<<numBlocks, threadsPerBlock>>>(
      numPeople, currentTime, trafficPersonVec_d, edgesData_d, laneMap_d,
      intersections_d, deltaTime);
  gpuErrchk(hipPeekAtLastError());
  peopleBench.stopMeasuring();
  //    if (random_bool(gen)){
  //        peopleBench.startMeasuring();
  //        // Simulate people.
  //        kernel_trafficSimulation<<<numBlocks, threadsPerBlock>>>(
  //                numPeople, currentTime, trafficPersonVec_d, edgesData_d,
  //                laneMap_d,
  //                        intersections_d, deltaTime);
  //        gpuErrchk(hipPeekAtLastError());
  //        peopleBench.stopMeasuring();
  //
  //        // Simulate intersections.
  //        intersectionBench.startMeasuring();
  //        kernel_intersectionOneSimulation<<<numBlocks, threadsPerBlock>>>(
  //                numIntersections, edgesData_d, intersections_d,
  //                trafficPersonVec_d,
  //                        laneMap_d);
  //        gpuErrchk(hipPeekAtLastError());
  //        intersectionBench.stopMeasuring();
  //    }
  //    else{
  //        // Simulate intersections.
  //        intersectionBench.startMeasuring();
  //        kernel_intersectionOneSimulation<<<numBlocks, threadsPerBlock>>>(
  //                numIntersections, edgesData_d, intersections_d,
  //                trafficPersonVec_d,
  //                        laneMap_d);
  //        gpuErrchk(hipPeekAtLastError());
  //        intersectionBench.stopMeasuring();
  //
  //        peopleBench.startMeasuring();
  //        // Simulate people.
  //        kernel_trafficSimulation<<<numBlocks, threadsPerBlock>>>(
  //                numPeople, currentTime, trafficPersonVec_d, edgesData_d,
  //                laneMap_d,
  //                        intersections_d, deltaTime);
  //        gpuErrchk(hipPeekAtLastError());
  //        peopleBench.stopMeasuring();
  //
  //    }

} //
