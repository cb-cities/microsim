#include "hip/hip_runtime.h"
// CUDA CODE
#include "assert.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>

#include "agent.h"
#include "b18EdgeData.h"
#include <iostream>
#include <vector>

#include "config.h"
#include "src/benchmarker.h"

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

///////////////////////////////
// CONSTANTS

__constant__ float intersectionClearance = 7.8f; // TODO(pavan): WHAT IS THIS?

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n",
           hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double)free_byte;
  double total_db = (double)total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n",
         used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0,
         total_db / 1024.0 / 1024.0);
}
////////////////////////////////
// VARIABLES
LC::Agent *trafficPersonVec_d;
uint *indexPathVec_d;
LC::B18EdgeData *edgesData_d;

__constant__ bool calculatePollution = true;
__constant__ float cellSize = 1.0f;

//__constant__ float deltaTime = 0.5f;
// const float deltaTimeH = 0.5f;

// const uint numStepsPerSample = 30.0f / deltaTimeH; //each min
// const uint numStepsTogether = 12; //change also in density (10 per hour)

uchar *laneMap_d;
bool readFirstMapC = true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
float startTime;

LC::B18IntersectionData *intersections_d;
uchar *trafficLights_d;

float *accSpeedPerLinePerTimeInterval_d;
float *numVehPerLinePerTimeInterval_d;

void b18InitCUDA(
    bool fistInitialization, std::vector<LC::Agent> &trafficPersonVec,
    std::vector<uint> &indexPathVec, std::vector<LC::B18EdgeData> &edgesData,
    std::vector<uchar> &laneMap, std::vector<uchar> &trafficLights,
    std::vector<LC::B18IntersectionData> &intersections, float startTimeH,
    float endTimeH, std::vector<float> &accSpeedPerLinePerTimeInterval,
    std::vector<float> &numVehPerLinePerTimeInterval, float deltaTime) {
  // printf(">>b18InitCUDA firstInitialization %s\n",
  // (fistInitialization?"INIT":"ALREADY INIT")); printMemoryUsage();

  const uint numStepsPerSample = 30.0f / deltaTime; // each min
  const uint numStepsTogether = 12; // change also in density (10 per hour)
  {                                 // people
    size_t size = trafficPersonVec.size() * sizeof(LC::Agent);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&trafficPersonVec_d,
                           size)); // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, trafficPersonVec.data(), size,
                         hipMemcpyHostToDevice));
  }

  { // indexPathVec
    size_t sizeIn = indexPathVec.size() * sizeof(uint);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&indexPathVec_d,
                           sizeIn)); // Allocate array on device
    gpuErrchk(hipMemcpy(indexPathVec_d, indexPathVec.data(), sizeIn,
                         hipMemcpyHostToDevice));
  }
  { // edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::B18EdgeData);
    if (fistInitialization)
      gpuErrchk(
          hipMalloc((void **)&edgesData_d, sizeD)); // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD,
                         hipMemcpyHostToDevice));
  }
  { // laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization)
      gpuErrchk(
          hipMalloc((void **)&laneMap_d, sizeL)); // Allocate array on device
    gpuErrchk(
        hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  { // intersections
    size_t sizeI = intersections.size() * sizeof(LC::B18IntersectionData);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&intersections_d,
                           sizeI)); // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, intersections.data(), sizeI,
                         hipMemcpyHostToDevice));
    size_t sizeT = trafficLights.size() * sizeof(uchar); // total number of
                                                         // lanes
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&trafficLights_d,
                           sizeT)); // Allocate array on device
    gpuErrchk(hipMemcpy(trafficLights_d, trafficLights.data(), sizeT,
                         hipMemcpyHostToDevice));
  }
  {
    startTime = startTimeH * 3600.0f;
    uint numSamples =
        ceil(((endTimeH * 3600.0f - startTimeH * 3600.0f) /
              (deltaTime * numStepsPerSample * numStepsTogether))) +
        1; //!!!
    accSpeedPerLinePerTimeInterval.clear();
    numVehPerLinePerTimeInterval.clear();
    accSpeedPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    numVehPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    size_t sizeAcc = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&accSpeedPerLinePerTimeInterval_d,
                           sizeAcc)); // Allocate array on device
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&numVehPerLinePerTimeInterval_d,
                           sizeAcc)); // Allocate array on device
    gpuErrchk(hipMemset(&accSpeedPerLinePerTimeInterval_d[0], 0, sizeAcc));
    gpuErrchk(hipMemset(&numVehPerLinePerTimeInterval_d[0], 0, sizeAcc));
  }
  printMemoryUsage();
} //

void b18FinishCUDA(void) {
  //////////////////////////////
  // FINISH
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
  hipFree(trafficLights_d);

  hipFree(accSpeedPerLinePerTimeInterval_d);
  hipFree(numVehPerLinePerTimeInterval_d);
} //

void b18GetDataCUDA(std::vector<LC::Agent> &trafficPersonVec,
                    std::vector<LC::B18EdgeData> &edgesData,
                    std::vector<LC::B18IntersectionData> &intersections) {
  // copy back people
  size_t size = trafficPersonVec.size() * sizeof(LC::Agent);
  size_t size_edges = edgesData.size() * sizeof(LC::B18EdgeData);
  size_t size_intersections =
      intersections.size() * sizeof(LC::B18IntersectionData);

  hipMemcpy(trafficPersonVec.data(), trafficPersonVec_d, size,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
  hipMemcpy(edgesData.data(), edgesData_d, size_edges,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
  hipMemcpy(intersections.data(), intersections_d, size_intersections,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
}

__device__ uint lanemap_pos(const uint currentEdge, const uint edge_length,
                            const uint laneNum, const uint pos_in_lane) {
  uint kMaxMapWidthM = 1024;
  uint num_cell = pos_in_lane / kMaxMapWidthM;
  int tot_num_cell = edge_length / kMaxMapWidthM;
  if (edge_length % kMaxMapWidthM) {
    tot_num_cell += 1;
  }
  return kMaxMapWidthM * currentEdge + kMaxMapWidthM * laneNum * tot_num_cell +
         kMaxMapWidthM * num_cell + pos_in_lane % kMaxMapWidthM;
}

__device__ void calculateGaps(uint mapToReadShift, uchar *laneMap,
                              LC::Agent &agent, uint laneToCheck, float &gap_a,
                              float &gap_b, uchar &v_a, uchar &v_b) {

  int kMaxMapWidthM = 1024;
  // CHECK FORWARD
  for (ushort b = agent.posInLaneM - 1; b < agent.length;
       b++) { // NOTE -1 to make sure there is none in at the same level
    auto posToSample =
        lanemap_pos(agent.currentEdge, agent.length, laneToCheck, b);
    if (laneMap[mapToReadShift + posToSample] != 0xFF) {
      gap_a = b - agent.posInLaneM; // m
      v_a = laneMap[mapToReadShift + posToSample] / 3;
      break;
    }
  }
  // CHECK BACKWARD
  for (ushort b = agent.posInLaneM + 1; b > 0;
       b--) { // NOTE -1 to make sure there is none in at the same level
    auto posToSample =
        lanemap_pos(agent.currentEdge, agent.length, laneToCheck, b);
    if (laneMap[mapToReadShift + posToSample] != 0xFF) {
      gap_b = agent.posInLaneM - b; // m
      v_b = laneMap[mapToReadShift + posToSample] / 3;
      break;
    }
  }
}
__device__ void calculateGapsLC(uint mapToReadShift, uchar *laneMap,
                                uchar trafficLightState, uint laneToCheck,
                                uint currentEdge, float posInMToCheck,
                                float length, uchar &v_a, uchar &v_b,
                                float &gap_a, float &gap_b) {

  ushort numOfCells = ceil(length);
  ushort initShift = ceil(posInMToCheck);
  uchar laneChar;
  bool found = false;
  int kMaxMapWidthM = 1024;

  // CHECK FORWARD
  // printf("initShift %u numOfCells %u\n",initShift,numOfCells);
  for (ushort b = initShift - 1; (b < numOfCells);
       b++) { // NOTE -1 to make sure there is none in at the same level
    // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
    const uint posToSample =
        mapToReadShift + kMaxMapWidthM * (currentEdge + laneToCheck) + b;
    laneChar = laneMap[posToSample];

    if (laneChar != 0xFF) {
      gap_a = ((float)b - initShift); // m
      v_a = laneChar; // laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }

  if (!found) {
    if (trafficLightState == 0x00) { // red
      // found=true;
      gap_a = gap_b = 1000.0f; // force to change to the line without vehicle
      v_a = v_b = 0xFF;
      return;
    }
  }

  if (!found) {
    gap_a = 1000.0f;
  }

  // CHECK BACKWARDS
  found = false;
  // printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
  for (int b = initShift + 1; (b >= 0);
       b--) { // NOTE +1 to make sure there is none in at the same level
    // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
    const uint posToSample =
        mapToReadShift + kMaxMapWidthM * (currentEdge + laneToCheck) + b;
    laneChar = laneMap[posToSample];
    if (laneChar != 0xFF) {
      gap_b = ((float)initShift - b); // m
      v_b = laneChar; // laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }

  // printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
  if (!found) {
    gap_b = 1000.0f;
  }

} //

__device__ void calculateLaneCarShouldBe(uint curEdgeLane, uint nextEdge,
                                         LC::B18IntersectionData *intersections,
                                         uint edgeNextInters,
                                         ushort edgeNumLanes,
                                         ushort &initOKLanes,
                                         ushort &endOKLanes) {

  initOKLanes = 0;
  endOKLanes = edgeNumLanes;
  bool currentEdgeFound = false;
  bool exitFound = false;
  ushort numExitToTake = 0;
  ushort numExists = 0;

  for (int eN = intersections[edgeNextInters].totalInOutEdges - 1; eN >= 0;
       eN--) { // clockwise
    uint procEdge = intersections[edgeNextInters].edge[eN];

    if ((procEdge & kMaskLaneMap) == curEdgeLane) { // current edge 0xFFFFF
      currentEdgeFound = true;
      if (exitFound == false) {
        numExitToTake = 0;
      }
      continue;
    }

    if ((procEdge & kMaskInEdge) == 0x0) { // out edge 0x800000
      numExists++;
      if (currentEdgeFound == true) {
        numExitToTake++;
      }
      if (currentEdgeFound == false && exitFound == false) {
        numExitToTake++;
      }
    }
    if ((procEdge & kMaskInEdge) == nextEdge) {
      exitFound = true;
      currentEdgeFound = false;
    }
  }

  if (edgeNumLanes == 0) {
    printf("ERRRROR\n");
  }

  switch (edgeNumLanes) {
  /// ONE LANE
  case 1:
    initOKLanes = 0;
    endOKLanes = 1;
    break;

    /// TWO LANE
  case 2:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = 2;
      break;

    case 3:
      if (numExitToTake > 2) { // left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;

    default:
      if (numExitToTake >= numExists - 1) {
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;
    }

    break;

    /// THREE LANE
  case 3:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = 3;
      break;

    case 3:
      if (numExitToTake > 2) { // left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 3;
      break;

    default:
      if (numExitToTake >= numExists - 1) {
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;
    }

    break;

  case 4:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = 4;
      break;

    case 3:
      if (numExitToTake == 1) { // right
        initOKLanes = 3;
        endOKLanes = 4;
      }

      if (numExitToTake > 3) { // left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 4;
      break;

    default:
      if (numExitToTake == 1) { // right
        initOKLanes = edgeNumLanes - 1;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake >= numExists - 2) {
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1; // also lane 2
      endOKLanes = edgeNumLanes;
    }

    break;

  default:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = edgeNumLanes;
      break;

    case 3:
      if (numExitToTake == 1) { // right
        initOKLanes = edgeNumLanes - 1;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake > edgeNumLanes - 2) { // left
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1;
      endOKLanes = edgeNumLanes;
      break;

    default:
      if (numExitToTake < 2) { // right
        initOKLanes = edgeNumLanes - 2;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake >= numExists - 2) {
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1; // also lane 2
      endOKLanes = edgeNumLanes - 1;
    }

    break;
  }
} //

__device__ void initialize_agent(LC::Agent &agent, LC::B18EdgeData *edgesData,
                                 uint *indexPathVec, uchar *laneMap,
                                 uint mapToReadShift, uint mapToWriteShift) {

  // 1.2 find first edge
  agent.indexPathCurr = agent.indexPathInit; // reset index.
  agent.currentEdge = indexPathVec[agent.indexPathCurr];
  agent.nextEdge = indexPathVec[agent.indexPathCurr + 1];
  if (agent.currentEdge == -1) {
    agent.active = 2;
    // printf("0xFFFF\n");
    return;
  }

  // 1.3 update person edgeData
  // COPY DATA FROM EDGE TO PERSON
  agent.edgeNumLanes = edgesData[agent.currentEdge].numLines;
  agent.edgeNextInters = edgesData[agent.currentEdge].nextIntersMapped;
  agent.length = edgesData[agent.currentEdge].length;

  // printf("edgesData length %f\n",edgesData[firstEdge].length);
  agent.maxSpeedMperSec = edgesData[agent.currentEdge].maxSpeedMperSec;
  // printf("edgesData %.10f\n",edgesData[firstEdge].maxSpeedMperSec);

  // 1.4 try to place the car
  ushort lN = agent.edgeNumLanes - 1;
  bool enough_space = true;
  for (auto b = 0; b < agent.s_0; b++) {
    // just right LANE !!!!!!!
    auto pos = lanemap_pos(agent.currentEdge, agent.length, lN, b);
    auto laneChar =
        laneMap[mapToReadShift + pos]; // get byte of edge (proper line)
    if (laneChar != 0xFF) {
      enough_space = false;
      break;
    }
  }
  if (enough_space) {
    agent.v = 0;
    agent.LC_stateofLaneChanging = 0;
    agent.numOfLaneInEdge = lN;
    agent.posInLaneM = 0; // m
    uchar vInMpS =
        (uchar)(agent.v * 3); // speed in m/s *3 (to keep more precisio
    auto pos = lanemap_pos(agent.currentEdge, agent.length,
                           agent.numOfLaneInEdge, agent.posInLaneM);
    laneMap[mapToWriteShift + pos] = vInMpS;
    atomicAdd(
        &(edgesData[indexPathVec[agent.indexPathCurr]].upstream_veh_count), 1);
  } else {
    agent.num_steps++;
    agent.waited_steps++;
    return;
  }

  // 1.5 active car

  agent.active = 1;
  agent.isInIntersection = 0;
  // trafficPersonVec[p].nextPathEdge++;//incremet so it continues in next
  // edge

  // 1.6 update next edge
  if (agent.nextEdge != -1) {
    agent.nextEdgemaxSpeedMperSec = edgesData[agent.nextEdge].maxSpeedMperSec;
    agent.nextEdgeNumLanes = edgesData[agent.nextEdge].numLines;
    agent.nextEdgeNextInters = edgesData[agent.nextEdge].nextIntersMapped;
    agent.nextEdgeLength = edgesData[agent.nextEdge].length;
    // trafficPersonVec[p].nextPathEdge++;
    agent.LC_initOKLanes = 0xFF;
    agent.LC_endOKLanes = 0xFF;
  }
}

__device__ void check_front_car(LC::Agent &agent, uchar *laneMap,
                                float deltaTime, uint mapToReadShift) {

  int numCellsCheck = fmax(15.0f, agent.v * deltaTime); // 15 or speed*time
  ushort byteInLine = (ushort)floor(agent.posInLaneM);
  ushort numOfCells = ceil((agent.length) - 2);

  // a) SAME LINE (BEFORE SIGNALING)
  bool found = false;
  float s = 30;
  float delta_v = agent.v - agent.maxSpeedMperSec;
  for (ushort b = byteInLine + 2; (b < numOfCells) && (numCellsCheck > 0);
       b++, numCellsCheck--) {

    uint posToSample =
        lanemap_pos(agent.currentEdge, agent.length, agent.numOfLaneInEdge, b);
    auto laneChar = laneMap[mapToReadShift + posToSample];
    if (laneChar != 0xFF) {
      s = ((float)(b - byteInLine)); // m
      delta_v =
          agent.v -
          (laneChar / 3.0f); // laneChar is in 3*ms (to save space in array)
      found = true;
      agent.thirdTerm = b;
      break;
    }
  }
  // NEXT LINE
  // e) MOVING ALONG IN THE NEXT EDGE
  //  if (!found && numCellsCheck > 0) { // check if in next line
  //    if ((agent.nextEdge != -1)) {    // we haven't arrived to
  //      // destination next line)
  //      ushort nextEdgeLaneToBe = agent.numOfLaneInEdge; // same lane
  //
  //      // printf("trafficPersonVec[p].numOfLaneInEdge
  //      // %u\n",trafficPersonVec[p].numOfLaneInEdge);
  //      if (nextEdgeLaneToBe >= agent.nextEdgeNumLanes) {
  //        nextEdgeLaneToBe =
  //            agent.nextEdgeNumLanes - 1; // change line if there are less
  //            roads
  //      }
  //
  //      ushort numOfCells = ceil(agent.nextEdgeLength);
  //
  //      for (ushort b = 0; (b < numOfCells) && (numCellsCheck > 0);
  //           b++, numCellsCheck--) {
  //        // laneChar = laneMap[mapToReadShift + maxWidth * (nextEdge +
  //        // nextEdgeLaneToBe) + b];
  //        uint posToSample = lanemap_pos(agent.nextEdge, nextEdgeLaneToBe, b);
  //        auto laneChar = laneMap[mapToReadShift + posToSample];
  //
  //        if (laneChar != 0xFF) {
  //          s = ((float)(b)); // m
  //          delta_v = agent.v -
  //                    (laneChar / 3.0f); // laneChar is in 3*ms (to save space
  //                    in
  //          break;
  //        }
  //      }
  //    }
  //  }

  agent.s = s;
  agent.delta_v = delta_v;
}

__device__ void update_agent_info(LC::Agent &agent, float deltaTime) {

  // update speed
  float thirdTerm = 0;
  if (agent.delta_v > 1) { // car in front and slower than us
    // 2.1.2 calculate dv_dt
    float s_star =
        agent.s_0 +
        fmax(0.0f, (agent.v * agent.T + (agent.v * agent.delta_v) /
                                            (2 * sqrtf(agent.a * agent.b))));

    thirdTerm = powf(((s_star) / (agent.s)), 2);
    agent.slow_down_steps++;
    // printf("s_star[%d] = %f\n", p, s_star);
    // printf(">FOUND s_star %f thirdTerm %f!!!!\n",s_star,thirdTerm);
  }
  float numMToMove;
  if (agent.v == 0 and agent.delta_v == 0 and agent.posInLaneM > 0 and
      agent.s < 5) {
    numMToMove = 0; // stopped at the middle
  } else {
    float dv_dt =
        agent.a *
        (1.0f - std::pow((agent.v / agent.maxSpeedMperSec), 4) - thirdTerm);
    agent.dv_dt = dv_dt;

    // 2.1.3 update values
    numMToMove =
        fmax(0.0f, agent.v * deltaTime + 0.5f * (dv_dt)*deltaTime * deltaTime);

    //    agent.thirdTerm = agent.v;
    agent.v += dv_dt * deltaTime;
    if (agent.v < 0) {
      agent.v = 0;
      numMToMove = 0;
    }
  }
  agent.m2move = numMToMove;
  agent.cum_length += numMToMove;
  agent.cum_v += agent.v;
  agent.posInLaneM += numMToMove;
}

__device__ void change_lane(LC::Agent &agent, uchar *laneMap,
                            uint mapToReadShift, uchar *trafficLights) {

  if (agent.posInLaneM > agent.length) { // skip if will go to next edge
    return;
  }
  if (agent.edgeNumLanes < 2 || agent.nextEdge == -1 ||
      agent.v > 0.9 * agent.maxSpeedMperSec) {
    return; // skip if reach the destination/have no lane to change/cruising
            // (avoid periodic lane changing)
  }

  if (agent.v > 3.0f &&           // at least 10km/h to try to change lane
      agent.delta_v > -0.1 &&     // decelerating or stuck
      agent.num_steps % 5 == 0) { // just check every (5 steps) 5 seconds

    bool leftLane = agent.numOfLaneInEdge > 0; // at least one lane on the left
    bool rightLane =
        agent.numOfLaneInEdge < agent.edgeNumLanes - 1; // at least one lane

    if (leftLane && rightLane) {
      if (int(agent.v) % 2 == 0) { // pseudo random for change lane
        leftLane = false;
      }
    }

    ushort laneToCheck = agent.numOfLaneInEdge - 1;
    if (rightLane) {
      laneToCheck = agent.numOfLaneInEdge + 1;
    }

    uchar v_a, v_b;
    float gap_a = 1000.0f, gap_b = 1000.0f;
    calculateGaps(mapToReadShift, laneMap, agent, laneToCheck, gap_a, gap_b,
                  v_a, v_b);
    // printf("p %u LC 1 %u\n",p,laneToCheck);
    //        uchar trafficLightState =
    //            trafficLights[agent.currentEdge + agent.numOfLaneInEdge];

    //        calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
    //        laneToCheck,
    //                        agent.currentEdge, agent.posInLaneM,
    //                        agent.length, v_a, v_b, gap_a, gap_b);
    // Safe distance calculation
    float b1A = 0.05, b2A = 0.15;
    float b1B = 0.15, b2B = 0.40;
    // simParameters.s_0-> critical lead gap
    float g_na_D =
        fmax(agent.s_0, agent.s_0 + b1A * agent.v + b2A * (agent.v - v_a));
    float g_bn_D =
        fmax(agent.s_0, agent.s_0 + b1B * v_b + b2B * (v_b - agent.v));
    if (gap_b < g_bn_D || gap_a < g_na_D) { // gap smaller than critical gap
      return;
    }

    agent.numOfLaneInEdge = laneToCheck; // CHANGE LINE
    agent.num_lane_change += 1;
  }
}

__device__ uint find_intersetcion_id(LC::Agent &agent,
                                     LC::B18EdgeData *edgesData) {
  // find the intersection id
  auto &current_edge = edgesData[agent.currentEdge];
  auto &next_edge = edgesData[agent.nextEdge];
  for (unsigned i = 0; i < 2; i++) {
    auto vid = current_edge.vertex[i];
    for (unsigned j = 0; j < 2; j++) {
      if (next_edge.vertex[j] == vid) {
        return vid;
      }
    }
  }
  return 0;
}

__device__ uint find_queue_id(LC::Agent &agent,
                              LC::B18IntersectionData &intersection) {
  for (unsigned i = 0; i < intersection.num_queue; i++) {
    if (agent.currentEdge == intersection.start_edge[i] and
        agent.nextEdge == intersection.end_edge[i]) {
      return i;
    }
  }
  //  int pos0, pos1;
  //  for (unsigned i = 0; i < intersection.num_edge; i++) {
  //    if (agent.currentEdge == intersection.lanemap_id[i]) {
  //      pos0 = i;
  //    }
  //    if (agent.nextEdge == intersection.lanemap_id[i]) {
  //      pos1 = i;
  //    }
  //  }
  //  int base_idx = 0;
  //  if (pos0 > pos1) {
  //    base_idx += intersection.num_queue;
  //    int temp = pos1;
  //    pos1 = pos0;
  //    pos0 = temp;
  //  }
  //  int idx = (pos0 * intersection.num_edge - (pos0 * (pos0 + 1)) / 2 + pos1 -
  //             pos0 - 1);
  //  return idx;
}

__shared__ int mutex;

__device__ bool update_intersection(int agent_id, LC::Agent &agent,
                                    LC::B18EdgeData *edgesData,
                                    LC::B18IntersectionData *intersections) {
  if (agent.posInLaneM < agent.length) { // does not reach an intersection
    return false;
  }
  if (agent.nextEdge == -1) { // reach destination
    agent.active = 2;
    atomicAdd(&(edgesData[agent.currentEdge].downstream_veh_count), 1);
    return false;
  }
  auto intersetcion_id = find_intersetcion_id(agent, edgesData);
  auto &intersection = intersections[intersetcion_id];
  int queue_id = find_queue_id(agent, intersection);
  auto &queue = intersection.queue[queue_id];
  auto &queue_ptr = intersection.pos[queue_id];
  agent.in_queue = true;
  agent.v = 0; // in queue vehicle is stopped.
  // Synchronization Control
  bool isSet = false;
  do {
    if (isSet = atomicCAS(&mutex, 0, 1) == 0) {
      queue[queue_ptr] = agent_id;
      atomicAdd(&(queue_ptr), 1);
      atomicAdd(&(edgesData[agent.currentEdge].downstream_veh_count), 1);
    }
    if (isSet) {
      atomicExch(&mutex, 0);
      __syncthreads();
    }
  } while (!isSet);
  return true;
}

__device__ void write2lane_map(LC::Agent &agent, LC::B18EdgeData *edgesData,
                               uint *indexPathVec, uchar *laneMap,
                               uint mapToWriteShift) {
  // write to the lanemap if still on the edge
  auto posToSample = lanemap_pos(agent.currentEdge, agent.length,
                                 agent.numOfLaneInEdge, agent.posInLaneM);
  uchar vInMpS = (uchar)(agent.v * 3); // speed in m/s to fit in uchar
  laneMap[mapToWriteShift + posToSample] = vInMpS;
}

// Kernel that executes on the CUDA device
__global__ void kernel_trafficSimulation(
    int numPeople, float currentTime, uint mapToReadShift, uint mapToWriteShift,
    LC::Agent *trafficPersonVec, uint *indexPathVec, LC::B18EdgeData *edgesData,
    uchar *laneMap, LC::B18IntersectionData *intersections,
    uchar *trafficLights, float deltaTime, const IDMParameters simParameters) {

  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p >= numPeople) {
    return; // CUDA check (inside margins)
  }
  if (threadIdx.x == 0) {
    mutex = 0;
  }
  __syncthreads();

  auto &agent = trafficPersonVec[p];
  // 1. initialization
  if (agent.active == 2) { // agent is already finished
    return;
  }
  // 1.1. check if person should still wait or should start
  if (agent.active == 0) {
    if (agent.time_departure > currentTime) { // wait
      return;
    } else { // its your turn
      initialize_agent(agent, edgesData, indexPathVec, laneMap, mapToReadShift,
                       mapToWriteShift);
      return;
    }
  }

  // 2. Moving
  agent.num_steps++;
  agent.nextEdge = indexPathVec[agent.indexPathCurr + 1];
  if (agent.in_queue) {
    agent.num_steps_in_queue += 1;
    return;
  }

  // 2.1.1 Find front car
  check_front_car(agent, laneMap, deltaTime, mapToReadShift);
  // 2.1.2 Update agent information using the front car info
  update_agent_info(agent, deltaTime);
  //  2.1.3 Perform lane changing if necessary
  change_lane(agent, laneMap, mapToReadShift, trafficLights);
  // 2.14 check intersection
  bool added2queue = update_intersection(p, agent, edgesData, intersections);
  //  // 2.1.4 write the updated agent info to lanemap
  if (not added2queue) {
    write2lane_map(agent, edgesData, indexPathVec, laneMap, mapToWriteShift);
  }

} //

/*
__global__ void kernel_intersectionSTOPSimulation(
     uint numIntersections,
     float currentTime,
     LC::B18IntersectionData *intersections,
     uchar *trafficLights,
     LC::B18EdgeData* edgesData,//for the length
     uchar* laneMap,//to check if there are cars
     uint mapToReadShift) {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i<numIntersections) {//CUDA check (inside margins)

     const float deltaEvent = 0.0f;

     //if(i==0)printf("i %d\n",i);
     if (currentTime > intersections[i].nextEvent &&
intersections[i].totalInOutEdges > 0) { uint edgeOT =
intersections[i].edge[intersections[i].state]; uchar numLinesO = edgeOT >> 24;
       uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF

       // red old traffic lights
       for (int nL = 0; nL < numLinesO; nL++) {
         trafficLights[edgeONum + nL] = 0x00; //red old traffic light
       }

       for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1; iN++) {
//to give a round intersections[i].state = (intersections[i].state + 1) %
           intersections[i].totalInOutEdges;//next light

         if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) ==
kMaskInEdge) {  // 0x800000 uint edgeIT =
intersections[i].edge[intersections[i].state]; uint edgeINum = edgeIT &
kMaskLaneMap; //get edgeI 0xFFFFF uchar numLinesI = edgeIT >> 24;
           /// check if someone in this edge
           int rangeToCheck = 5.0f; //5m
           ushort firstPosToCheck = edgesData[edgeINum].length -
intersectionClearance; //last po bool atLeastOneStopped = false;

           for (int posCheck = firstPosToCheck; rangeToCheck >= 0 && posCheck
>= 0; posCheck--, rangeToCheck--) { //as many cells as the rangeToCheck says
for (int nL = 0; nL < numLinesI; nL++) {
               //int cellNum = mapToReadShift + maxWidth * (edgeINum + nL) +
posCheck; const uint posToSample = mapToReadShift + kMaxMapWidthM * (edgeINum
+
(((int) (posCheck / kMaxMapWidthM)) * numLinesI) + nL) + posCheck %
kMaxMapWidthM;


               if (laneMap[posToSample] == 0) { //car stopped
                 trafficLights[edgeINum + nL] = 0x0F; // STOP SIGN 0x0F--> Let
pass atLeastOneStopped = true;
               }
             }
           }

           if (atLeastOneStopped == true) {
             intersections[i].nextEvent = currentTime + deltaEvent; //just
move forward time if changed (otherwise check in next iteration) break;
           }
         }
       }
     }
     ///
   }

}//
*/
__device__ bool check_space(int space, int eid, int edge_length, uchar *laneMap,
                            uint mapToReadShift) {
  for (auto b = 0; b < space; b++) {
    // just right LANE !!!!!!!
    auto pos = lanemap_pos(eid, edge_length, 0, b);
    auto laneChar =
        laneMap[mapToReadShift + pos]; // get byte of edge (proper line)
    if (laneChar != 0xFF) {
      return false;
    }
  }
  return true;
}

__device__ void move2nextEdge(LC::Agent &agent, int numMToMove,
                              LC::B18EdgeData *edgesData, uint *indexPathVec,
                              uchar *laneMap, uint mapToWriteShift) {

  agent.indexPathCurr++;
  agent.maxSpeedMperSec = agent.nextEdgemaxSpeedMperSec;
  agent.edgeNumLanes = agent.nextEdgeNumLanes;
  agent.edgeNextInters = agent.nextEdgeNextInters;
  agent.length = agent.nextEdgeLength;
  agent.posInLaneM = numMToMove;
  agent.currentEdge = indexPathVec[agent.indexPathCurr];
  atomicAdd(&(edgesData[agent.currentEdge].upstream_veh_count), 1);
  if (agent.numOfLaneInEdge >= agent.edgeNumLanes) {
    agent.numOfLaneInEdge =
        agent.edgeNumLanes - 1; // change line if there are less roads
  }
  ////////////
  // update next edge
  uint nextNEdge = indexPathVec[agent.indexPathCurr + 1];
  agent.nextEdge = nextNEdge;
  if (nextNEdge != -1) {
    // trafficPersonVec[p].nextPathEdge++;
    agent.LC_initOKLanes = 0xFF;
    agent.LC_endOKLanes = 0xFF;
    // 2.2.3 update person edgeData
    // trafficPersonVec[p].nextEdge=nextEdge;
    agent.nextEdgemaxSpeedMperSec = edgesData[nextNEdge].maxSpeedMperSec;
    agent.nextEdgeNumLanes = edgesData[nextNEdge].numLines;
    agent.nextEdgeNextInters = edgesData[nextNEdge].nextIntersMapped;
    agent.nextEdgeLength = edgesData[nextNEdge].length;
  }
  //
  agent.LC_stateofLaneChanging = 0;
  auto posToSample = lanemap_pos(agent.currentEdge, agent.length,
                                 agent.numOfLaneInEdge, agent.posInLaneM);
  uchar vInMpS = (uchar)(agent.v * 3); // speed in m/s to fit in uchar
  laneMap[mapToWriteShift + posToSample] = vInMpS;
  agent.in_queue = false;
}
__device__ bool empty_queue(LC::B18IntersectionData &intersection,
                            int queue_ptr, LC::Agent *trafficPersonVec,
                            LC::B18EdgeData *edgesData, uint *indexPathVec,
                            uchar *laneMap, uint mapToReadShift,
                            uint mapToWriteShift) {
  auto &q1 = intersection.queue[queue_ptr];
  auto &n1 = intersection.pos[queue_ptr];
  unsigned eid1 = intersection.end_edge[queue_ptr];
  int numMToMove1 = (n1 + 1) * 3;
  int edge_length = edgesData[eid1].length;
  bool enough_space1 = check_space(numMToMove1 + 6, eid1, edge_length, laneMap,
                                   mapToReadShift); // check 6m ahead
  intersection.max_queue = max(intersection.max_queue, n1);
  if (enough_space1) {
    for (int i = 0; i < n1; ++i) {
      auto &agent = trafficPersonVec[q1[i]];
      move2nextEdge(agent, numMToMove1, edgesData, indexPathVec, laneMap,
                    mapToWriteShift); // move to the next edge
      numMToMove1 -= 3;
    }
    intersection.pos[queue_ptr] = 0; // cleared, reset the pointer
    return true;
  }
  return false;
}

__device__ void place_stop(LC::Agent &agent, LC::B18EdgeData *edgesData,
                           uchar *laneMap, uint mapToWriteShift) {
  auto &edge = edgesData[agent.currentEdge];
  for (int j = 0; j < 5; ++j) {
    auto pos = agent.length - j;
    for (int i = 0; i < edge.numLines; ++i) {
      auto posToSample = lanemap_pos(agent.currentEdge, agent.length, i, pos);
      laneMap[mapToWriteShift + posToSample] = 0;
    }
  }
}

__global__ void kernel_intersectionOneSimulation(
    uint numIntersections, uint mapToWriteShift, uint mapToReadShift,
    LC::B18EdgeData *edgesData, LC::B18IntersectionData *intersections,
    uint *indexPathVec, LC::Agent *trafficPersonVec, uchar *laneMap) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numIntersections) {
    return; // CUDA check (inside margins)
  }
  auto &intersection = intersections[i];
  auto &queues = intersection.queue;
  auto &queue_counter = intersection.pos;
  unsigned start_ptr = intersection.queue_ptr;
  unsigned end_ptr = intersection.num_queue / 2;
  while (intersection.queue_ptr + 1 != start_ptr) {
    unsigned n1 = queue_counter[intersection.queue_ptr];
    unsigned n2 = queue_counter[end_ptr + intersection.queue_ptr];
    if (n1 + n2 > 0) {
      bool empty1 = empty_queue(intersection, intersection.queue_ptr,
                                trafficPersonVec, edgesData, indexPathVec,
                                laneMap, mapToReadShift, mapToWriteShift);
      bool empty2 = empty_queue(intersection, end_ptr + intersection.queue_ptr,
                                trafficPersonVec, edgesData, indexPathVec,
                                laneMap, mapToReadShift, mapToWriteShift);
      //
      if (!empty1 or !empty2) {
        if (intersection.queue_ptr == 0) {
          intersection.queue_ptr = end_ptr - 1;
        } else {
          intersection.queue_ptr -= 1;
        }
      } // if not clear, try it again for the next time
      break;
    }
    if (intersection.queue_ptr + 1 < end_ptr) {
      intersection.queue_ptr += 1;
    } else {
      intersection.queue_ptr = 0;
      if (start_ptr == 0)
        break;
    }
  }
  //     add stop sign for full queues
  for (unsigned j = 0; j < intersection.num_queue; j++) {
    auto num_cars = queue_counter[j];
    if (num_cars > 3) {
      auto &q1 = intersection.queue[j];
      auto &agent = trafficPersonVec[q1[0]];
      auto eid = intersection.start_edge[j];
      agent.located_eid = eid;
      place_stop(agent, edgesData, laneMap, mapToWriteShift);
    }
  }
}

//  if (i < numIntersections) {       // CUDA check (inside margins)
//    const float deltaEvent = 20.0f; /// !!!!
//    if (currentTime > intersections[i].nextEvent &&
//        intersections[i].totalInOutEdges > 0) {
//
//      uint edgeOT = intersections[i].edge[intersections[i].state];
//      uchar numLinesO = edgeOT >> 24;
//      uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF;
//
//      // red old traffic lights
//      if ((edgeOT & kMaskInEdge) ==
//          kMaskInEdge) { // Just do it if we were in in
//        for (int nL = 0; nL < numLinesO; nL++) {
//          trafficLights[edgeONum + nL] = 0x00; // red old traffic light
//        }
//      }
//
//      for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1;
//           iN++) { // to give a round
//        intersections[i].state = (intersections[i].state + 1) %
//                                 intersections[i].totalInOutEdges; // next
//                                 light
//
//        if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) ==
//            kMaskInEdge) { // 0x800000
//          // green new traffic lights
//          uint edgeIT = intersections[i].edge[intersections[i].state];
//          uint edgeINum = edgeIT & kMaskLaneMap; //  0xFFFFF; //get edgeI
//          uchar numLinesI = edgeIT >> 24;
//
//          for (int nL = 0; nL < numLinesI; nL++) {
//            trafficLights[edgeINum + nL] = 0xFF;
//          }
//
//          // trafficLights[edgeINum]=0xFF;
//          break;
//        }
//      } // green new traffic light
//
//      intersections[i].nextEvent = currentTime + deltaEvent;
//    }
//    //////////////////////////////////////////////////////
//  }

//} //

// Kernel that executes on the CUDA device
__global__ void kernel_sampleTraffic(
    int numPeople, LC::Agent *trafficPersonVec, uint *indexPathVec,
    float *accSpeedPerLinePerTimeInterval,
    float *numVehPerLinePerTimeInterval, // this could have been int
    uint offset) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {                     // CUDA check (inside margins)
    if (trafficPersonVec[p].active == 1) { // just active
      int edgeNum = indexPathVec[trafficPersonVec[p].indexPathCurr];
      accSpeedPerLinePerTimeInterval[edgeNum + offset] +=
          trafficPersonVec[p].v / 3.0f;
      numVehPerLinePerTimeInterval[edgeNum + offset]++;
    }
  }
}
__global__ void kernel_resetPeople(int numPeople, LC::Agent *trafficPersonVec) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) { // CUDA check (inside margins)
    trafficPersonVec[p].active = 0;
  }
}

void b18GetSampleTrafficCUDA(std::vector<float> &accSpeedPerLinePerTimeInterval,
                             std::vector<float> &numVehPerLinePerTimeInterval) {
  // copy back people
  size_t size = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
  hipMemcpy(accSpeedPerLinePerTimeInterval.data(),
             accSpeedPerLinePerTimeInterval_d, size, hipMemcpyDeviceToHost);

  size_t sizeI = numVehPerLinePerTimeInterval.size() * sizeof(uchar);
  hipMemcpy(numVehPerLinePerTimeInterval.data(),
             numVehPerLinePerTimeInterval_d, sizeI, hipMemcpyDeviceToHost);
}

void b18ResetPeopleLanesCUDA(uint numPeople) {
  kernel_resetPeople<<<ceil(numPeople / 1024.0f), 1024>>>(numPeople,
                                                          trafficPersonVec_d);
  hipMemset(&laneMap_d[0], -1, halfLaneMap * sizeof(unsigned char));
  hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap * sizeof(unsigned char));
}

void b18SimulateTrafficCUDA(float currentTime, uint numPeople,
                            uint numIntersections, float deltaTime,
                            const IDMParameters simParameters, int numBlocks,
                            int threadsPerBlock) {
  intersectionBench.startMeasuring();
  const uint numStepsTogether = 12; // change also in density (10 per hour)
  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if (readFirstMapC) {
    mapToReadShift = 0;
    mapToWriteShift = halfLaneMap;
    gpuErrchk(
        hipMemset(&laneMap_d[halfLaneMap], -1,
                   halfLaneMap * sizeof(unsigned char))); // clean second half
  } else {
    mapToReadShift = halfLaneMap;
    mapToWriteShift = 0;
    gpuErrchk(
        hipMemset(&laneMap_d[0], -1,
                   halfLaneMap * sizeof(unsigned char))); // clean first half
  }
  readFirstMapC = !readFirstMapC; // next iteration invert use

  // Simulate intersections.
  kernel_intersectionOneSimulation<<<numBlocks, threadsPerBlock>>>(
      numIntersections, mapToWriteShift, mapToReadShift, edgesData_d,
      intersections_d, indexPathVec_d, trafficPersonVec_d, laneMap_d);
  gpuErrchk(hipPeekAtLastError());

  intersectionBench.stopMeasuring();

  peopleBench.startMeasuring();
  // Simulate people.
  kernel_trafficSimulation<<<numBlocks, threadsPerBlock>>>(
      numPeople, currentTime, mapToReadShift, mapToWriteShift,
      trafficPersonVec_d, indexPathVec_d, edgesData_d, laneMap_d,
      intersections_d, trafficLights_d, deltaTime, simParameters);
  gpuErrchk(hipPeekAtLastError());
  peopleBench.stopMeasuring();

  // Sample if necessary.
  //  if ((((float)((int)currentTime)) == (currentTime)) &&
  //      ((int)currentTime % ((int)30)) == 0) { // 3min //(sample double each
  //      3min)
  //    int samplingNumber = (currentTime - startTime) / (30 *
  //    numStepsTogether); uint offset = numIntersections * samplingNumber;
  //    // printf("Sample %d\n", samplingNumber);
  //    kernel_sampleTraffic<<<ceil(numPeople / 1024.0f), 1024>>>(
  //        numPeople, trafficPersonVec_d, indexPathVec_d,
  //        accSpeedPerLinePerTimeInterval_d, numVehPerLinePerTimeInterval_d,
  //        offset);
  //    gpuErrchk(hipPeekAtLastError());
  //  }
} //
