#include "hip/hip_runtime.h"
// CUDA CODE
#include "assert.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>

#include "agent.h"
#include "b18EdgeData.h"
#include <iostream>
#include <vector>

#include "config.h"
#include "src/benchmarker.h"

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

///////////////////////////////
// CONSTANTS

__constant__ float intersectionClearance = 7.8f; // TODO(pavan): WHAT IS THIS?

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n",
           hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double)free_byte;
  double total_db = (double)total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n",
         used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0,
         total_db / 1024.0 / 1024.0);
}
////////////////////////////////
// VARIABLES
LC::Agent *trafficPersonVec_d;
uint *indexPathVec_d;
LC::B18EdgeData *edgesData_d;

__constant__ bool calculatePollution = true;
__constant__ float cellSize = 1.0f;

//__constant__ float deltaTime = 0.5f;
// const float deltaTimeH = 0.5f;

// const uint numStepsPerSample = 30.0f / deltaTimeH; //each min
// const uint numStepsTogether = 12; //change also in density (10 per hour)

uchar *laneMap_d;
bool readFirstMapC = true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
float startTime;

LC::B18IntersectionData *intersections_d;
uchar *trafficLights_d;

float *accSpeedPerLinePerTimeInterval_d;
float *numVehPerLinePerTimeInterval_d;

void b18InitCUDA(
    bool fistInitialization, std::vector<LC::Agent> &trafficPersonVec,
    std::vector<uint> &indexPathVec, std::vector<LC::B18EdgeData> &edgesData,
    std::vector<uchar> &laneMap, std::vector<uchar> &trafficLights,
    std::vector<LC::B18IntersectionData> &intersections, float startTimeH,
    float endTimeH, std::vector<float> &accSpeedPerLinePerTimeInterval,
    std::vector<float> &numVehPerLinePerTimeInterval, float deltaTime) {
  // printf(">>b18InitCUDA firstInitialization %s\n",
  // (fistInitialization?"INIT":"ALREADY INIT")); printMemoryUsage();

  const uint numStepsPerSample = 30.0f / deltaTime; // each min
  const uint numStepsTogether = 12; // change also in density (10 per hour)
  {                                 // people
    size_t size = trafficPersonVec.size() * sizeof(LC::Agent);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&trafficPersonVec_d,
                           size)); // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, trafficPersonVec.data(), size,
                         hipMemcpyHostToDevice));
  }

  { // indexPathVec
    size_t sizeIn = indexPathVec.size() * sizeof(uint);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&indexPathVec_d,
                           sizeIn)); // Allocate array on device
    gpuErrchk(hipMemcpy(indexPathVec_d, indexPathVec.data(), sizeIn,
                         hipMemcpyHostToDevice));
  }
  { // edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::B18EdgeData);
    if (fistInitialization)
      gpuErrchk(
          hipMalloc((void **)&edgesData_d, sizeD)); // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD,
                         hipMemcpyHostToDevice));
  }
  { // laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization)
      gpuErrchk(
          hipMalloc((void **)&laneMap_d, sizeL)); // Allocate array on device
    gpuErrchk(
        hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  { // intersections
    size_t sizeI = intersections.size() * sizeof(LC::B18IntersectionData);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&intersections_d,
                           sizeI)); // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, intersections.data(), sizeI,
                         hipMemcpyHostToDevice));
    size_t sizeT = trafficLights.size() * sizeof(uchar); // total number of
                                                         // lanes
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&trafficLights_d,
                           sizeT)); // Allocate array on device
    gpuErrchk(hipMemcpy(trafficLights_d, trafficLights.data(), sizeT,
                         hipMemcpyHostToDevice));
  }
  {
    startTime = startTimeH * 3600.0f;
    uint numSamples =
        ceil(((endTimeH * 3600.0f - startTimeH * 3600.0f) /
              (deltaTime * numStepsPerSample * numStepsTogether))) +
        1; //!!!
    accSpeedPerLinePerTimeInterval.clear();
    numVehPerLinePerTimeInterval.clear();
    accSpeedPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    numVehPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    size_t sizeAcc = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&accSpeedPerLinePerTimeInterval_d,
                           sizeAcc)); // Allocate array on device
    if (fistInitialization)
      gpuErrchk(hipMalloc((void **)&numVehPerLinePerTimeInterval_d,
                           sizeAcc)); // Allocate array on device
    gpuErrchk(hipMemset(&accSpeedPerLinePerTimeInterval_d[0], 0, sizeAcc));
    gpuErrchk(hipMemset(&numVehPerLinePerTimeInterval_d[0], 0, sizeAcc));
  }
  printMemoryUsage();
} //

void b18FinishCUDA(void) {
  //////////////////////////////
  // FINISH
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
  hipFree(trafficLights_d);

  hipFree(accSpeedPerLinePerTimeInterval_d);
  hipFree(numVehPerLinePerTimeInterval_d);
} //

void b18GetDataCUDA(std::vector<LC::Agent> &trafficPersonVec,
                    std::vector<LC::B18EdgeData> &edgesData) {
  // copy back people
  size_t size = trafficPersonVec.size() * sizeof(LC::Agent);
  size_t size_edges = edgesData.size() * sizeof(LC::B18EdgeData);
  hipMemcpy(trafficPersonVec.data(), trafficPersonVec_d, size,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
  hipMemcpy(edgesData.data(), edgesData_d, size_edges,
             hipMemcpyDeviceToHost); // hipMemcpyHostToDevice
}

__device__ void calculateGapsLC(uint mapToReadShift, uchar *laneMap,
                                uchar trafficLightState, uint laneToCheck,
                                uint currentEdge, float posInMToCheck,
                                float length, uchar &v_a, uchar &v_b,
                                float &gap_a, float &gap_b) {

  ushort numOfCells = ceil(length);
  ushort initShift = ceil(posInMToCheck);
  uchar laneChar;
  bool found = false;
  int kMaxMapWidthM = 1024;

  // CHECK FORWARD
  // printf("initShift %u numOfCells %u\n",initShift,numOfCells);
  for (ushort b = initShift - 1; (b < numOfCells);
       b++) { // NOTE -1 to make sure there is none in at the same level
    // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
    const uint posToSample =
        mapToReadShift + kMaxMapWidthM * (currentEdge + laneToCheck) + b;
    laneChar = laneMap[posToSample];

    if (laneChar != 0xFF) {
      gap_a = ((float)b - initShift); // m
      v_a = laneChar; // laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }

  if (!found) {
    if (trafficLightState == 0x00) { // red
      // found=true;
      gap_a = gap_b = 1000.0f; // force to change to the line without vehicle
      v_a = v_b = 0xFF;
      return;
    }
  }

  if (!found) {
    gap_a = 1000.0f;
  }

  // CHECK BACKWARDS
  found = false;
  // printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
  for (int b = initShift + 1; (b >= 0);
       b--) { // NOTE +1 to make sure there is none in at the same level
    // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
    const uint posToSample =
        mapToReadShift + kMaxMapWidthM * (currentEdge + laneToCheck) + b;
    laneChar = laneMap[posToSample];
    if (laneChar != 0xFF) {
      gap_b = ((float)initShift - b); // m
      v_b = laneChar; // laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }

  // printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
  if (!found) {
    gap_b = 1000.0f;
  }

} //

__device__ void calculateLaneCarShouldBe(uint curEdgeLane, uint nextEdge,
                                         LC::B18IntersectionData *intersections,
                                         uint edgeNextInters,
                                         ushort edgeNumLanes,
                                         ushort &initOKLanes,
                                         ushort &endOKLanes) {

  initOKLanes = 0;
  endOKLanes = edgeNumLanes;
  bool currentEdgeFound = false;
  bool exitFound = false;
  ushort numExitToTake = 0;
  ushort numExists = 0;

  for (int eN = intersections[edgeNextInters].totalInOutEdges - 1; eN >= 0;
       eN--) { // clockwise
    uint procEdge = intersections[edgeNextInters].edge[eN];

    if ((procEdge & kMaskLaneMap) == curEdgeLane) { // current edge 0xFFFFF
      currentEdgeFound = true;
      if (exitFound == false) {
        numExitToTake = 0;
      }
      continue;
    }

    if ((procEdge & kMaskInEdge) == 0x0) { // out edge 0x800000
      numExists++;
      if (currentEdgeFound == true) {
        numExitToTake++;
      }
      if (currentEdgeFound == false && exitFound == false) {
        numExitToTake++;
      }
    }
    if ((procEdge & kMaskInEdge) == nextEdge) {
      exitFound = true;
      currentEdgeFound = false;
    }
  }

  if (edgeNumLanes == 0) {
    printf("ERRRROR\n");
  }

  switch (edgeNumLanes) {
  /// ONE LANE
  case 1:
    initOKLanes = 0;
    endOKLanes = 1;
    break;

    /// TWO LANE
  case 2:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = 2;
      break;

    case 3:
      if (numExitToTake > 2) { // left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;

    default:
      if (numExitToTake >= numExists - 1) {
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;
    }

    break;

    /// THREE LANE
  case 3:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = 3;
      break;

    case 3:
      if (numExitToTake > 2) { // left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 3;
      break;

    default:
      if (numExitToTake >= numExists - 1) {
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;
    }

    break;

  case 4:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = 4;
      break;

    case 3:
      if (numExitToTake == 1) { // right
        initOKLanes = 3;
        endOKLanes = 4;
      }

      if (numExitToTake > 3) { // left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 4;
      break;

    default:
      if (numExitToTake == 1) { // right
        initOKLanes = edgeNumLanes - 1;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake >= numExists - 2) {
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1; // also lane 2
      endOKLanes = edgeNumLanes;
    }

    break;

  default:
    switch (numExists) {
    case 1:
    case 2: // all okay
      initOKLanes = 0;
      endOKLanes = edgeNumLanes;
      break;

    case 3:
      if (numExitToTake == 1) { // right
        initOKLanes = edgeNumLanes - 1;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake > edgeNumLanes - 2) { // left
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1;
      endOKLanes = edgeNumLanes;
      break;

    default:
      if (numExitToTake < 2) { // right
        initOKLanes = edgeNumLanes - 2;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake >= numExists - 2) {
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1; // also lane 2
      endOKLanes = edgeNumLanes - 1;
    }

    break;
  }
} //

__device__ void initialize_agent(LC::Agent *trafficPersonVec, int p,
                                 LC::B18EdgeData *edgesData, uint *indexPathVec,
                                 uchar *laneMap,
                                 const IDMParameters simParameters,
                                 uint mapToReadShift, uint mapToWriteShift) {
  int kMaxMapWidthM = 1024;
  // start
  // printf("p %d edge = %u\n", p, trafficPersonVec[p].indexPathInit);
  // 1.2 find first edge
  trafficPersonVec[p].indexPathCurr =
      trafficPersonVec[p].indexPathInit; // reset index.
  uint firstEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
  uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
  // printf("indexPathVec %d = %u nextEdge = %u\n", p,
  // indexPathVec[trafficPersonVec[p].indexPathCurr],
  // indexPathVec[trafficPersonVec[p].indexPathCurr + 1]);

  if (firstEdge == -1) {
    trafficPersonVec[p].active = 2;
    // printf("0xFFFF\n");
    return;
  }

  // 1.3 update person edgeData
  // COPY DATA FROM EDGE TO PERSON
  trafficPersonVec[p].edgeNumLanes = edgesData[firstEdge].numLines;
  trafficPersonVec[p].edgeNextInters = edgesData[firstEdge].nextIntersMapped;
  // printf("edgeNextInters %u = %u\n", firstEdge,
  // edgesData[firstEdge].nextIntersMapped);

  trafficPersonVec[p].length = edgesData[firstEdge].length;

  // printf("edgesData length %f\n",edgesData[firstEdge].length);
  trafficPersonVec[p].maxSpeedMperSec = edgesData[firstEdge].maxSpeedMperSec;
  // printf("edgesData %.10f\n",edgesData[firstEdge].maxSpeedMperSec);

  // 1.4 try to place the car
  ushort lN = trafficPersonVec[p].edgeNumLanes - 1;
  bool enough_space = true;
  for (auto b = 0; b < simParameters.s_0; b++) {
    // just right LANE !!!!!!!
    auto laneChar = laneMap[mapToReadShift + kMaxMapWidthM * (firstEdge + lN) +
                            b]; // get byte of edge (proper line)
    if (laneChar != 0xFF) {
      enough_space = false;
      break;
    }
  }
  if (enough_space) {
    trafficPersonVec[p].v = 0;
    trafficPersonVec[p].LC_stateofLaneChanging = 0;
    trafficPersonVec[p].numOfLaneInEdge = lN;
    trafficPersonVec[p].posInLaneM = 0; // m
    uchar vInMpS = (uchar)(trafficPersonVec[p].v *
                           3); // speed in m/s *3 (to keep more precision
    laneMap[mapToWriteShift + kMaxMapWidthM * (firstEdge + lN)] = vInMpS;
    atomicAdd(&(edgesData[indexPathVec[trafficPersonVec[p].indexPathCurr]]
                    .upstream_veh_count),
              1);
  } else {
    trafficPersonVec[p].num_steps++;
    trafficPersonVec[p].waited_steps++;
    return;
  }

  // 1.5 active car

  trafficPersonVec[p].active = 1;
  trafficPersonVec[p].isInIntersection = 0;
  trafficPersonVec[p].co = 0.0f;
  trafficPersonVec[p].gas = 0.0f;
  // trafficPersonVec[p].nextPathEdge++;//incremet so it continues in next
  // edge

  // trafficPersonVec[p].nextEdge=nextEdge;
  if (nextEdge != -1) {
    trafficPersonVec[p].nextEdgemaxSpeedMperSec =
        edgesData[nextEdge].maxSpeedMperSec;
    trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdge].numLines;
    trafficPersonVec[p].nextEdgeNextInters =
        edgesData[nextEdge].nextIntersMapped;
    trafficPersonVec[p].nextEdgeLength = edgesData[nextEdge].length;
    // trafficPersonVec[p].nextPathEdge++;
    trafficPersonVec[p].LC_initOKLanes = 0xFF;
    trafficPersonVec[p].LC_endOKLanes = 0xFF;
  }
}

__device__ void check_front_car(LC::Agent *trafficPersonVec, int p,
                                uint *indexPathVec, uchar *laneMap,
                                float deltaTime, uint mapToReadShift,
                                float *front_car_info) {

  int kMaxMapWidthM = 1024;
  uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
  uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

  int numCellsCheck = fmax(50.0f, trafficPersonVec[p].v * deltaTime *
                                      2); // 30 or double of the speed*time
  ushort byteInLine = (ushort)floor(trafficPersonVec[p].posInLaneM);
  ushort numOfCells = ceil((trafficPersonVec[p].length) - 2);

  // a) SAME LINE (BEFORE SIGNALING)
  bool found = false;
  float s{0};
  float delta_v{0};
  for (ushort b = byteInLine + 2; (b < numOfCells) && (numCellsCheck > 0);
       b++, numCellsCheck--) {
    const uint posToSample =
        mapToReadShift +
        kMaxMapWidthM * (currentEdge + trafficPersonVec[p].numOfLaneInEdge) + b;
    auto laneChar = laneMap[posToSample];
    if (laneChar != 0xFF) {
      s = ((float)(b - byteInLine)); // m
      delta_v =
          trafficPersonVec[p].v -
          (laneChar / 3.0f); // laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }
  // NEXT LINE
  // e) MOVING ALONG IN THE NEXT EDGE
  if (!found && numCellsCheck > 0) { // check if in next line
    if ((nextEdge != -1) &&
        (trafficPersonVec[p].edgeNextInters !=
         trafficPersonVec[p].end_intersection)) { // we haven't arrived to
      // destination next line)
      ushort nextEdgeLaneToBe =
          trafficPersonVec[p].numOfLaneInEdge; // same lane

      // printf("trafficPersonVec[p].numOfLaneInEdge
      // %u\n",trafficPersonVec[p].numOfLaneInEdge);
      if (nextEdgeLaneToBe >= trafficPersonVec[p].nextEdgeNumLanes) {
        nextEdgeLaneToBe = trafficPersonVec[p].nextEdgeNumLanes -
                           1; // change line if there are less roads
      }

      // printf("2trafficPersonVec[p].numOfLaneInEdge
      // %u\n",trafficPersonVec[p].numOfLaneInEdge);
      ushort numOfCells = ceil(trafficPersonVec[p].nextEdgeLength);

      for (ushort b = 0; (b < numOfCells) && (numCellsCheck > 0);
           b++, numCellsCheck--) {
        // laneChar = laneMap[mapToReadShift + maxWidth * (nextEdge +
        // nextEdgeLaneToBe) + b];
        const uint posToSample = mapToReadShift +
                                 kMaxMapWidthM * (nextEdge + nextEdgeLaneToBe) +
                                 b; // b18 not changed since we check
        auto laneChar = laneMap[posToSample];

        if (laneChar != 0xFF) {
          s = ((float)(b)); // m
          delta_v = trafficPersonVec[p].v -
                    (laneChar / 3.0f); // laneChar is in 3*ms (to save space in
          break;
        }
      }
    }
  }
  front_car_info[0] = s;
  front_car_info[1] = delta_v;
}

__device__ void update_agent_info(LC::Agent *trafficPersonVec, int p,
                                  float deltaTime,
                                  const IDMParameters simParameters,
                                  float *front_car_info) {
  auto s = front_car_info[0];
  auto delta_v = front_car_info[1];

  // update speed
  float thirdTerm = 0;
  if (delta_v > 1) { // car in front and slower than us
    // if (found == true) { //car in front and slower than us
    // 2.1.2 calculate dv_dt
    float s_star =
        simParameters.s_0 +
        fmax(0.0f,
             (trafficPersonVec[p].v * trafficPersonVec[p].T +
              (trafficPersonVec[p].v * delta_v) /
                  (2 * sqrtf(trafficPersonVec[p].a * trafficPersonVec[p].b))));
    thirdTerm = powf(((s_star) / (s)), 2);
    trafficPersonVec[p].slow_down_steps++;
    // printf("s_star[%d] = %f\n", p, s_star);
    // printf(">FOUND s_star %f thirdTerm %f!!!!\n",s_star,thirdTerm);
  }

  float dv_dt =
      trafficPersonVec[p].a *
      (1.0f -
       std::pow((trafficPersonVec[p].v / trafficPersonVec[p].maxSpeedMperSec),
                4) -
       thirdTerm);

  // 2.1.3 update values
  auto numMToMove = fmax(0.0f, trafficPersonVec[p].v * deltaTime +
                                   0.5f * (dv_dt)*deltaTime * deltaTime);
  trafficPersonVec[p].cum_length += numMToMove;
  trafficPersonVec[p].v += dv_dt * deltaTime;
  if (trafficPersonVec[p].v < 0) {
    trafficPersonVec[p].v = 0;
    dv_dt = 0.0f;
  }
  trafficPersonVec[p].cum_v += trafficPersonVec[p].v;
  trafficPersonVec[p].posInLaneM = trafficPersonVec[p].posInLaneM + numMToMove;
}

__device__ void change_lane(LC::Agent *trafficPersonVec, int p,
                            const IDMParameters simParameters,
                            uint *indexPathVec, uchar *laneMap,
                            uint mapToReadShift,
                            uchar *trafficLights) {

  if (trafficPersonVec[p].posInLaneM >
      trafficPersonVec[p].length) { // skip if will go to next edge
    return;
  }

  uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
  uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
  if (trafficPersonVec[p].edgeNumLanes < 1 && nextEdge == -1) {
    return; // skip if reach the end or have no lane to change
  }

  if (trafficPersonVec[p].v > 3.0f && // at least 10km/h to try to change lane
      trafficPersonVec[p].num_steps % 5 ==
          0) { // just check every (5 steps) 5 seconds and make sure the agent
               // has enough speed
    // LC 1 update lane changing status
    if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
      // 2.2-exp((x-1)^2)
      float x = trafficPersonVec[p].posInLaneM / trafficPersonVec[p].length;

      if (x > 0.4f) { // just after 40% of the road
        float probabiltyMandatoryState = 2.2 - exp((x - 1) * (x - 1));

        // if (((float) qrand() / RAND_MAX) < probabiltyMandatoryState) {
        if ((((int)(x * 100) % 100) / 100.0f) <
            probabiltyMandatoryState) { // pseudo random number
          trafficPersonVec[p].LC_stateofLaneChanging = 1;
        }
      }
    }

    ////////////////////////////////////////////////////
    // LC 2 NOT MANDATORY STATE
    if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
      // discretionary change: v slower than the current road limit and
      // deccelerating and moving
      if ((trafficPersonVec[p].v <
           (trafficPersonVec[p].maxSpeedMperSec * 0.8f)) &&
          trafficPersonVec[p].v > 3.0f) {
        bool leftLane = trafficPersonVec[p].numOfLaneInEdge >
                        0; // at least one lane on the left
        bool rightLane =
            trafficPersonVec[p].numOfLaneInEdge <
            trafficPersonVec[p].edgeNumLanes - 1; // at least one lane

        if (leftLane && rightLane) {
          if (int(trafficPersonVec[p].v) % 2 ==
              0) { // pseudo random for change lane
            leftLane = false;
          }
        }

        ushort laneToCheck;
        if (leftLane) {
          laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
        } else {
          laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
        }

        uchar v_a, v_b;
        float gap_a, gap_b;
        // printf("p %u LC 1 %u\n",p,laneToCheck);
        uchar trafficLightState =
            trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge];

        calculateGapsLC(mapToReadShift, laneMap, trafficLightState, laneToCheck,
                        currentEdge, trafficPersonVec[p].posInLaneM,
                        trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

        if (gap_a == 1000.0f && gap_b == 1000.0f) { // lag and lead car very far
          trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE

        } else { // NOT ALONE
          float b1A = 0.05f, b2A = 0.15f;
          float b1B = 0.15f, b2B = 0.40f;
          // simParameters.s_0-> critical lead gap
          float g_na_D, g_bn_D;
          bool acceptLC = true;

          if (gap_a != 1000.0f) {
            g_na_D = fmax(simParameters.s_0,
                          simParameters.s_0 + b1A * trafficPersonVec[p].v +
                              b2A * (trafficPersonVec[p].v - v_a * 3.0f));

            if (gap_a < g_na_D) { // gap smaller than critical gap
              acceptLC = false;
            }
          }

          if (acceptLC && gap_b != 1000.0f) {
            g_bn_D = fmax(simParameters.s_0,
                          simParameters.s_0 + b1B * v_b * 3.0f +
                              b2B * (v_b * 3.0f - trafficPersonVec[p].v));

            if (gap_b < g_bn_D) { // gap smaller than critical gap
              acceptLC = false;
            }
          }

          if (acceptLC) {
            trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
          }
        }
      }

    } // Discretionary
  }
}

__device__ void write2lane_map(LC::Agent *trafficPersonVec, int p,
                               LC::B18EdgeData *edgesData, uint *indexPathVec,
                               uchar *laneMap, uint mapToWriteShift) {
  int kMaxMapWidthM = 1024;
  uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
  uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
  // write to the lanemap if still on the edge
  if (trafficPersonVec[p].posInLaneM <
      trafficPersonVec[p].length) { // does not reach an intersection
    uchar vInMpS =
        (uchar)(trafficPersonVec[p].v * 3); // speed in m/s to fit in uchar
    ushort posInLineCells = (ushort)(trafficPersonVec[p].posInLaneM);
    const uint posToSample =
        mapToWriteShift +
        kMaxMapWidthM * (currentEdge + trafficPersonVec[p].numOfLaneInEdge) +
        posInLineCells;
    laneMap[posToSample] = vInMpS;
    return;
  }
  // 2.2.1 find next edge
  auto numMToMove = trafficPersonVec[p].posInLaneM - trafficPersonVec[p].length;
  trafficPersonVec[p].dist_traveled += trafficPersonVec[p].length;
  atomicAdd(&(edgesData[indexPathVec[trafficPersonVec[p].indexPathCurr]]
                  .downstream_veh_count),
            1);

  if (nextEdge == -1) {             // if(curr_intersection==end_intersection)
    trafficPersonVec[p].active = 2; // finished
    return;
  }
  // move to the next edge
  trafficPersonVec[p].indexPathCurr++;
  trafficPersonVec[p].maxSpeedMperSec =
      trafficPersonVec[p].nextEdgemaxSpeedMperSec;
  trafficPersonVec[p].edgeNumLanes = trafficPersonVec[p].nextEdgeNumLanes;
  trafficPersonVec[p].edgeNextInters = trafficPersonVec[p].nextEdgeNextInters;
  trafficPersonVec[p].length = trafficPersonVec[p].nextEdgeLength;
  trafficPersonVec[p].posInLaneM = numMToMove;

  atomicAdd(&(edgesData[indexPathVec[trafficPersonVec[p].indexPathCurr]]
                  .upstream_veh_count),
            1);
  if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].edgeNumLanes) {
    trafficPersonVec[p].numOfLaneInEdge =
        trafficPersonVec[p].edgeNumLanes -
        1; // change line if there are less roads
  }

  ////////////
  // update next edge
  uint nextNEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
  // trafficPersonVec[p].nextEdge=nextEdge;
  if (nextNEdge != -1) {
    // trafficPersonVec[p].nextPathEdge++;
    trafficPersonVec[p].LC_initOKLanes = 0xFF;
    trafficPersonVec[p].LC_endOKLanes = 0xFF;

    // 2.2.3 update person edgeData
    // trafficPersonVec[p].nextEdge=nextEdge;
    trafficPersonVec[p].nextEdgemaxSpeedMperSec =
        edgesData[nextNEdge].maxSpeedMperSec;
    trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextNEdge].numLines;
    trafficPersonVec[p].nextEdgeNextInters =
        edgesData[nextNEdge].nextIntersMapped;
    trafficPersonVec[p].nextEdgeLength = edgesData[nextNEdge].length;
  }

  trafficPersonVec[p].LC_stateofLaneChanging = 0;
  uchar vInMpS =
      (uchar)(trafficPersonVec[p].v * 3); // speed in m/s to fit in uchar
  ushort posInLineCells = (ushort)(trafficPersonVec[p].posInLaneM);
  const uint posToSample =
      mapToWriteShift +
      kMaxMapWidthM * (currentEdge + trafficPersonVec[p].numOfLaneInEdge) +
      posInLineCells;
  laneMap[posToSample] = vInMpS;
}

// Kernel that executes on the CUDA device
__global__ void kernel_trafficSimulation(
    int numPeople, float currentTime, uint mapToReadShift, uint mapToWriteShift,
    LC::Agent *trafficPersonVec, uint *indexPathVec, LC::B18EdgeData *edgesData,
    uchar *laneMap, LC::B18IntersectionData *intersections,
    uchar *trafficLights, float deltaTime, const IDMParameters simParameters) {

  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p >= numPeople) {
    return; // CUDA check (inside margins)
  }

  // 1. initialization
  if (trafficPersonVec[p].active == 2) { // finished
    return;
  }
  // 1.1. check if person should still wait or should start
  if (trafficPersonVec[p].active == 0) {
    if (trafficPersonVec[p].time_departure > currentTime) { // wait
      return;
    } else { // its your turn
      initialize_agent(trafficPersonVec, p, edgesData, indexPathVec, laneMap,
                       simParameters, mapToReadShift, mapToWriteShift);
      return;
    }
  }

  // 2. Moving
  trafficPersonVec[p].num_steps++;
  trafficPersonVec[p].nextEdge =
      indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

  // 2.1.1 Find front car
  float front_car_info[] = {0, 0};
  check_front_car(trafficPersonVec, p, indexPathVec, laneMap, deltaTime,
                  mapToReadShift, front_car_info);

  // 2.1.2 Update agent information using the front car info
  update_agent_info(trafficPersonVec, p, deltaTime, simParameters,
                    front_car_info);

  // 2.1.3 Perform lane changing if necessary
//  change_lane(trafficPersonVec, p, simParameters, indexPathVec, laneMap,
//              mapToReadShift, trafficLights);
  // 2.1.4 write the updated agent info to lanemap
  write2lane_map(trafficPersonVec, p, edgesData, indexPathVec, laneMap,
                 mapToWriteShift);

} //

/*
__global__ void kernel_intersectionSTOPSimulation(
     uint numIntersections,
     float currentTime,
     LC::B18IntersectionData *intersections,
     uchar *trafficLights,
     LC::B18EdgeData* edgesData,//for the length
     uchar* laneMap,//to check if there are cars
     uint mapToReadShift) {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i<numIntersections) {//CUDA check (inside margins)

     const float deltaEvent = 0.0f;

     //if(i==0)printf("i %d\n",i);
     if (currentTime > intersections[i].nextEvent &&
intersections[i].totalInOutEdges > 0) { uint edgeOT =
intersections[i].edge[intersections[i].state]; uchar numLinesO = edgeOT >> 24;
       uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF

       // red old traffic lights
       for (int nL = 0; nL < numLinesO; nL++) {
         trafficLights[edgeONum + nL] = 0x00; //red old traffic light
       }

       for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1; iN++) {
//to give a round intersections[i].state = (intersections[i].state + 1) %
           intersections[i].totalInOutEdges;//next light

         if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) ==
kMaskInEdge) {  // 0x800000 uint edgeIT =
intersections[i].edge[intersections[i].state]; uint edgeINum = edgeIT &
kMaskLaneMap; //get edgeI 0xFFFFF uchar numLinesI = edgeIT >> 24;
           /// check if someone in this edge
           int rangeToCheck = 5.0f; //5m
           ushort firstPosToCheck = edgesData[edgeINum].length -
intersectionClearance; //last po bool atLeastOneStopped = false;

           for (int posCheck = firstPosToCheck; rangeToCheck >= 0 && posCheck
>= 0; posCheck--, rangeToCheck--) { //as many cells as the rangeToCheck says
for (int nL = 0; nL < numLinesI; nL++) {
               //int cellNum = mapToReadShift + maxWidth * (edgeINum + nL) +
posCheck; const uint posToSample = mapToReadShift + kMaxMapWidthM * (edgeINum
+
(((int) (posCheck / kMaxMapWidthM)) * numLinesI) + nL) + posCheck %
kMaxMapWidthM;


               if (laneMap[posToSample] == 0) { //car stopped
                 trafficLights[edgeINum + nL] = 0x0F; // STOP SIGN 0x0F--> Let
pass atLeastOneStopped = true;
               }
             }
           }

           if (atLeastOneStopped == true) {
             intersections[i].nextEvent = currentTime + deltaEvent; //just
move forward time if changed (otherwise check in next iteration) break;
           }
         }
       }
     }
     ///
   }

}//
*/

__global__ void
kernel_intersectionOneSimulation(uint numIntersections, float currentTime,
                                 LC::B18IntersectionData *intersections,
                                 uchar *trafficLights) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numIntersections) {       // CUDA check (inside margins)
    const float deltaEvent = 20.0f; /// !!!!
    if (currentTime > intersections[i].nextEvent &&
        intersections[i].totalInOutEdges > 0) {

      uint edgeOT = intersections[i].edge[intersections[i].state];
      uchar numLinesO = edgeOT >> 24;
      uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF;

      // red old traffic lights
      if ((edgeOT & kMaskInEdge) ==
          kMaskInEdge) { // Just do it if we were in in
        for (int nL = 0; nL < numLinesO; nL++) {
          trafficLights[edgeONum + nL] = 0x00; // red old traffic light
        }
      }

      for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1;
           iN++) { // to give a round
        intersections[i].state = (intersections[i].state + 1) %
                                 intersections[i].totalInOutEdges; // next light

        if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) ==
            kMaskInEdge) { // 0x800000
          // green new traffic lights
          uint edgeIT = intersections[i].edge[intersections[i].state];
          uint edgeINum = edgeIT & kMaskLaneMap; //  0xFFFFF; //get edgeI
          uchar numLinesI = edgeIT >> 24;

          for (int nL = 0; nL < numLinesI; nL++) {
            trafficLights[edgeINum + nL] = 0xFF;
          }

          // trafficLights[edgeINum]=0xFF;
          break;
        }
      } // green new traffic light

      intersections[i].nextEvent = currentTime + deltaEvent;
    }
    //////////////////////////////////////////////////////
  }

} //

// Kernel that executes on the CUDA device
__global__ void kernel_sampleTraffic(
    int numPeople, LC::Agent *trafficPersonVec, uint *indexPathVec,
    float *accSpeedPerLinePerTimeInterval,
    float *numVehPerLinePerTimeInterval, // this could have been int
    uint offset) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {                     // CUDA check (inside margins)
    if (trafficPersonVec[p].active == 1) { // just active
      int edgeNum = indexPathVec[trafficPersonVec[p].indexPathCurr];
      accSpeedPerLinePerTimeInterval[edgeNum + offset] +=
          trafficPersonVec[p].v / 3.0f;
      numVehPerLinePerTimeInterval[edgeNum + offset]++;
    }
  }
}
__global__ void kernel_resetPeople(int numPeople, LC::Agent *trafficPersonVec) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) { // CUDA check (inside margins)
    trafficPersonVec[p].active = 0;
  }
}

void b18GetSampleTrafficCUDA(std::vector<float> &accSpeedPerLinePerTimeInterval,
                             std::vector<float> &numVehPerLinePerTimeInterval) {
  // copy back people
  size_t size = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
  hipMemcpy(accSpeedPerLinePerTimeInterval.data(),
             accSpeedPerLinePerTimeInterval_d, size, hipMemcpyDeviceToHost);

  size_t sizeI = numVehPerLinePerTimeInterval.size() * sizeof(uchar);
  hipMemcpy(numVehPerLinePerTimeInterval.data(),
             numVehPerLinePerTimeInterval_d, sizeI, hipMemcpyDeviceToHost);
}

void b18ResetPeopleLanesCUDA(uint numPeople) {
  kernel_resetPeople<<<ceil(numPeople / 1024.0f), 1024>>>(numPeople,
                                                          trafficPersonVec_d);
  hipMemset(&laneMap_d[0], -1, halfLaneMap * sizeof(unsigned char));
  hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap * sizeof(unsigned char));
}

void b18SimulateTrafficCUDA(float currentTime, uint numPeople,
                            uint numIntersections, float deltaTime,
                            const IDMParameters simParameters, int numBlocks,
                            int threadsPerBlock) {
  intersectionBench.startMeasuring();
  const uint numStepsTogether = 12; // change also in density (10 per hour)
  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if (readFirstMapC == true) {
    mapToReadShift = 0;
    mapToWriteShift = halfLaneMap;
    gpuErrchk(
        hipMemset(&laneMap_d[halfLaneMap], -1,
                   halfLaneMap * sizeof(unsigned char))); // clean second half
  } else {
    mapToReadShift = halfLaneMap;
    mapToWriteShift = 0;
    gpuErrchk(
        hipMemset(&laneMap_d[0], -1,
                   halfLaneMap * sizeof(unsigned char))); // clean first half
  }
  readFirstMapC = !readFirstMapC; // next iteration invert use

  // Simulate intersections.
  kernel_intersectionOneSimulation<<<ceil(numIntersections / 512.0f), 512>>>(
      numIntersections, currentTime, intersections_d, trafficLights_d);
  gpuErrchk(hipPeekAtLastError());

  intersectionBench.stopMeasuring();

  peopleBench.startMeasuring();
  // Simulate people.
  kernel_trafficSimulation<<<numBlocks, threadsPerBlock>>>(
      numPeople, currentTime, mapToReadShift, mapToWriteShift,
      trafficPersonVec_d, indexPathVec_d, edgesData_d, laneMap_d,
      intersections_d, trafficLights_d, deltaTime, simParameters);
  gpuErrchk(hipPeekAtLastError());
  peopleBench.stopMeasuring();

  // Sample if necessary.
  if ((((float)((int)currentTime)) == (currentTime)) &&
      ((int)currentTime % ((int)30)) == 0) { // 3min //(sample double each 3min)
    int samplingNumber = (currentTime - startTime) / (30 * numStepsTogether);
    uint offset = numIntersections * samplingNumber;
    // printf("Sample %d\n", samplingNumber);
    kernel_sampleTraffic<<<ceil(numPeople / 1024.0f), 1024>>>(
        numPeople, trafficPersonVec_d, indexPathVec_d,
        accSpeedPerLinePerTimeInterval_d, numVehPerLinePerTimeInterval_d,
        offset);
    gpuErrchk(hipPeekAtLastError());
  }
} //
